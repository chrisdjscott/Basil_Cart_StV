#include "hip/hip_runtime.h"
﻿//////////////////////////////////////////////////////////////////////////////////
//						                                                        //
//Copyright (C) 2018 Bosserelle                                                 //
//                                                                              //
//This program is free software: you can redistribute it and/or modify          //
//it under the terms of the GNU General Public License as published by          //
//the Free Software Foundation.                                                 //
//                                                                              //
//This program is distributed in the hope that it will be useful,               //
//but WITHOUT ANY WARRANTY; without even the implied warranty of                //    
//MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the                 //
//GNU General Public License for more details.                                  //
//                                                                              //
//You should have received a copy of the GNU General Public License             //
//along with this program.  If not, see <http://www.gnu.org/licenses/>.         //
//////////////////////////////////////////////////////////////////////////////////

// includes, system


#include "Header.cuh"



//double phi = (1.0f + sqrt(5.0f)) / 2;
//double aphi = 1 / (phi + 1);
//double bphi = phi / (phi + 1);
//double twopi = 8 * atan(1.0f);
double epsilon = 1e-30;
//double g = 1.0;// 9.81;
//double rho = 1025.0;
//double eps = 0.0001;
//double CFL = 0.5;
//
//double totaltime = 0.0;
//
//
//double dt, dx;
//int nx, ny;
//
//double delta;

double *x, *y;
double *x_g, *y_g;

float *zs, *hh, *zb, *uu, *vv;//for CPU
double *zs_d, *hh_d, *zb_d, *uu_d, *vv_d; // double array only allocated instead of thge float if requested
float *zs_g, *hh_g, *zb_g, *uu_g, *vv_g; // for GPU
double *zs_gd, *hh_gd, *zb_gd, *uu_gd, *vv_gd;

float *zso, *hho, *uuo, *vvo;
double *zso_d, *hho_d, *uuo_d, *vvo_d;
float *zso_g, *hho_g, *uuo_g, *vvo_g; // for GPU
double *zso_gd, *hho_gd, *uuo_gd, *vvo_gd;
//CPU
float * dhdx, *dhdy, *dudx, *dudy, *dvdx, *dvdy;
float *dzsdx, *dzsdy;
//GPU
float * dhdx_g, *dhdy_g, *dudx_g, *dudy_g, *dvdx_g, *dvdy_g;
float *dzsdx_g, *dzsdy_g;
//double *fmu, *fmv;

double * dhdx_d, *dhdy_d, *dudx_d, *dudy_d, *dvdx_d, *dvdy_d;
double *dzsdx_d, *dzsdy_d;

double * dhdx_gd, *dhdy_gd, *dudx_gd, *dudy_gd, *dvdx_gd, *dvdy_gd;
double *dzsdx_gd, *dzsdy_gd;

float *Su, *Sv, *Fqux, *Fquy, *Fqvx, *Fqvy;
float * Fhu, *Fhv;
float * dh, *dhu, *dhv;

double *Su_d, *Sv_d, *Fqux_d, *Fquy_d, *Fqvx_d, *Fqvy_d;
double * Fhu_d, *Fhv_d;
double * dh_d, *dhu_d, *dhv_d;

//GPU
float *Su_g, *Sv_g, *Fqux_g, *Fquy_g, *Fqvx_g, *Fqvy_g;
float * Fhu_g, *Fhv_g;
float * dh_g, *dhu_g, *dhv_g;

double *Su_gd, *Sv_gd, *Fqux_gd, *Fquy_gd, *Fqvx_gd, *Fqvy_gd;
double * Fhu_gd, *Fhv_gd;
double * dh_gd, *dhu_gd, *dhv_gd;

float * TSstore, *TSstore_g;
double * TSstore_d, *TSstore_gd;

float * hhmean, *uumean, *vvmean, *zsmean;
float * hhmean_g, *uumean_g, *vvmean_g, *zsmean_g;
double * hhmean_d, *uumean_d, *vvmean_d, *zsmean_d;
double * hhmean_gd, *uumean_gd, *vvmean_gd, *zsmean_gd;

float * hhmax, *uumax, *vvmax, *zsmax;
float * hhmax_g, *uumax_g, *vvmax_g, *zsmax_g;
double * hhmax_d, *uumax_d, *vvmax_d, *zsmax_d;
double * hhmax_gd, *uumax_gd, *vvmax_gd, *zsmax_gd;

float * vort, *vort_g;// Vorticity output
double * vort_d, *vort_gd;

float dtmax = (float) (1.0 / epsilon);
double dtmax_d = 1.0 / epsilon;

double * dtmax_gd;
float * dtmax_g;

float *arrmax_g;
float *arrmin_g;
float *arrmin;

double *arrmax_gd;
double *arrmin_gd;
double *arrmin_d;

float * dummy;
double * dummy_d;
//std::string outfile = "output.nc";
//std::vector<std::string> outvars;
std::map<std::string, float *> OutputVarMapCPU;
std::map<std::string, double *> OutputVarMapCPUD;
std::map<std::string, float *> OutputVarMapGPU;
std::map<std::string, double *> OutputVarMapGPUD;
std::map<std::string, int> OutputVarMaplen;

hipArray* leftWLS_gp; // Cuda array to pre-store HD vel data before converting to textures
hipArray* rightWLS_gp;
hipArray* topWLS_gp;
hipArray* botWLS_gp;

hipChannelFormatDesc channelDescleftbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescrightbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDescbotbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
hipChannelFormatDesc channelDesctopbnd = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);

#include "Flow_kernel.cu"

void CUDA_CHECK(hipError_t CUDerr)
{


	if (hipSuccess != CUDerr) {

		fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n", \

			__FILE__, __LINE__, hipGetErrorString(CUDerr));

		exit(EXIT_FAILURE);

	}
}

unsigned int nextPow2(unsigned int x)
{
	--x;
	x |= x >> 1;
	x |= x >> 2;
	x |= x >> 4;
	x |= x >> 8;
	x |= x >> 16;
	return ++x;
}

template <class T> void Allocate1GPU(int nx, int ny, T *&zb_g)
{
	CUDA_CHECK(hipMalloc((void **)&zb_g, nx*ny * sizeof(T)));
}
template <class T> void Allocate4GPU(int nx, int ny, T *&zs_g, T *&hh_g, T *&uu_g, T *&vv_g)
{
	CUDA_CHECK(hipMalloc((void **)&zs_g, nx*ny * sizeof(T)));
	CUDA_CHECK(hipMalloc((void **)&hh_g, nx*ny * sizeof(T)));
	CUDA_CHECK(hipMalloc((void **)&uu_g, nx*ny * sizeof(T)));
	CUDA_CHECK(hipMalloc((void **)&vv_g, nx*ny * sizeof(T)));
}

template <class T> void Allocate1CPU(int nx, int ny, T *&zb)
{
	zb = (T *)malloc(nx*ny * sizeof(T));
}

template <class T> void Allocate4CPU(int nx, int ny, T *&zs, T *&hh, T *&uu, T *&vv)
{
	
	zs = (T *)malloc(nx*ny * sizeof(T));
	hh = (T *)malloc(nx*ny * sizeof(T));
	uu = (T *)malloc(nx*ny * sizeof(T));
	vv = (T *)malloc(nx*ny * sizeof(T));
}

template <class T> void setedges(int nx, int ny, T *&zb)
{
	for (int j = 0; j < ny; j++)
	{

		for (int i = 0; i < nx; i++)
		{
			if (i == 0)
			{
				zb[i + j*nx] = zb[(i + 1) + j*nx];
			}
			if (i == nx - 1)
			{
				zb[i + j*nx] = zb[(i - 1) + j*nx];
			}
			if (j == 0)
			{
				zb[i + j*nx] = zb[i + (j + 1)*nx];
			}
			if (j == ny - 1)
			{
				zb[i + j*nx] = zb[i + (j - 1)*nx];
			}

		}
	}
}

float maxdiff(int nxny, float * ref, float * pred)
{
	float maxd = 0.0f;
	for (int i = 0; i < nxny; i++)
	{
		maxd = max(abs(pred[i] - ref[i]), maxd);
	}
	return maxd;
}

float maxdiffID(int nx, int ny, int &im, int &jm,  float * ref, float * pred)
{
	float maxd = 0.0f;
	
	for (int i = 0; i < nx; i++)
	{
		for (int j = 0; j < ny; j++)
		{
			if (abs(pred[i] - ref[i]) > maxd)
			{
				im = i;
				jm = j;
				maxd = abs(pred[i] - ref[i]);
			}
		}
	}
	return maxd;
}

void checkloopGPU(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;
	

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	dim3 blockDimLine(32, 1, 1);
	dim3 gridDimLine(ceil((nx*ny*1.0f) / blockDimLine.x), 1, 1);

	

	float maxerr = 1e-11f;//1e-7f

	

	float maxdiffer;

	int imax = 0;
	int jmax = 0;

	dtmax = (float) (1.0 / epsilon);
	//float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0 >> > (nx, ny, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, dtmax_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	float mindtmax = 1.0f / 1e-30f;
	for (int i = 0; i < nx*ny; i++)
	{
		mindtmax = min(dummy[i], mindtmax);
	}


	//update step 1

	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, hh_g, dhdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, hh_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, zs_g, dzsdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, zs_g, dzsdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, uu_g, dudx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, uu_g, dudy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, vv_g, dvdx_g);
	CUDA_CHECK(hipDeviceSynchronize());

	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, vv_g, dvdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//update(int nx, int ny, double dt, double eps, double g, double CFL, double delta, float *hh, float *zs, float *uu, float *vv, float *&dh, float *&dhu, float *&dhv);
	update(nx, ny, (float)XParam.theta, (float)XParam.dt, (float)XParam.eps, (float)XParam.g, (float)XParam.CFL, (float)XParam.delta, hh, zs, uu, vv, dh, dhu, dhv);



	CUDA_CHECK(hipMemcpy(dummy, hh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, hh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdx: %f\n", maxdiffer);
	}


	
	// check gradients

	CUDA_CHECK(hipMemcpy(dummy, dhdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dhdx, dummy);
	if (maxdiffer > maxerr)
	{ 
		printf("High error in dhdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dhdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dzsdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dzsdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dudx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dudy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdy: %f\n", maxdiffer);
	}


	// All good so far continuing

	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, hh_g, zs_g, uu_g, vv_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, hh_g, zs_g, uu_g, vv_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, Fhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	//maxdiffer = maxdiff(nx*ny, Fhu, dummy);
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Fhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhu (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Fhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhv (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqux_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	
	
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Fqux, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqux (%f) in i=%d, j=%d\n", maxdiffer,imax,jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Fqvx, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvx (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Fqvy, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvy (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fquy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Fquy, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in Fquy (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Su_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Su, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in Su (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	CUDA_CHECK(hipMemcpy(dummy, Sv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, Sv, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in Sv (%f) in i=%d, j=%d\n", maxdiffer, imax, jmax);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////
	minmaxKernel << <gridDimLine, blockDimLine, 0 >> >(nx*ny, arrmax_g, arrmin_g, dtmax_g);
	//CUT_CHECK_ERROR("UpdateZom execution failed\n");
	CUDA_CHECK(hipDeviceSynchronize());

	finalminmaxKernel << <1, blockDimLine, 0 >> >(arrmax_g, arrmin_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipMemcpy(arrmax, arrmax_g, nx*ny*sizeof(DECNUM), hipMemcpyDeviceToHost));
	CUDA_CHECK(hipMemcpy(arrmin, arrmin_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));

	maxdiffer = abs(dtmax-arrmin[0]);


	CUDA_CHECK(hipMemcpy(dummy, dtmax_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));


	
	mindtmax=1.0f/1e-30f;
	for (int i = 0; i < nx*ny; i++)
	{
		mindtmax=min(dummy[i], mindtmax);
	}
	maxdiffer = abs(dtmax - mindtmax);

	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.delta, (float)XParam.g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, dh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dh, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in dh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dhu, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in dhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, dhv, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in dhv: %f\n", maxdiffer);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////
	XParam.dt = arrmin[0];
	
	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.dt*0.5f, (float)XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//predictor
	advance(nx, ny, (float)XParam.dt*0.5f, (float)XParam.eps,zb, hh, zs, uu, vv, dh, dhu, dhv, hho, zso, uuo, vvo);

	CUDA_CHECK(hipMemcpy(dummy, zso_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, zso, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in zso: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hho_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, hho, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in hho: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uuo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, uuo, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in uuo: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vvo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiffID(nx, ny, imax, jmax, vvo, dummy);;
	if (maxdiffer > maxerr)
	{
		printf("High error in vvo: %f\n", maxdiffer);
	}

	// All good so far continuing
	///////////////////////////////////////////////////////////////////////
	///////////////////////////////////////////////////////////////////////

	//corrector
	update(nx, ny, (float)XParam.theta, (float)XParam.dt, (float)XParam.eps, (float)XParam.g, (float)XParam.CFL, (float)XParam.delta, hho, zso, uuo, vvo, dh, dhu, dhv);

	//corrector setp
	//update again
	// calculate gradients
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, hho_g, dhdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, hho_g, dhdy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, zso_g, dzsdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, zso_g, dzsdy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, uuo_g, dudx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, uuo_g, dudy_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, vvo_g, dvdx_g);
	CUDA_CHECK(hipDeviceSynchronize());
	gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, vvo_g, dvdy_g);
	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	CUDA_CHECK(hipDeviceSynchronize());

	// check gradients

	CUDA_CHECK(hipMemcpy(dummy, dhdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dzsdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dzsdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dzsdy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dudy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dudy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dudy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dvdy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dvdy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dvdy: %f\n", maxdiffer);
	}



	updateKurgX << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());

	// no reduction of dtmax during the corrector step


	CUDA_CHECK(hipMemcpy(dummy, Fhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fhv: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqux_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqux, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqux: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvx_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvx, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvx: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fqvy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fqvy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fqvy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Fquy_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Fquy, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Fquy: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Su_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Su, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Su: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, Sv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, Sv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in Sv: %f\n", maxdiffer);
	}


	updateEV << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.delta, (float)XParam.g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());


	CUDA_CHECK(hipMemcpy(dummy, dh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, dhv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, dhv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in dhv: %f\n", maxdiffer);
	}


	advance(nx, ny, (float)XParam.dt, (float)XParam.eps,zb, hh, zs, uu, vv, dh, dhu, dhv, hho, zso, uuo, vvo);

	//
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.dt, (float)XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipMemcpy(dummy, zso_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, zso, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in zso: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hho_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hho, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in hho: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uuo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, uuo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in uuo: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vvo_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, vvo, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in vvo: %f\n", maxdiffer);
	}

	cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);

	cleanupGPU << <gridDim, blockDim, 0 >> >(nx, ny, hho_g, zso_g, uuo_g, vvo_g, hh_g, zs_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());



	CUDA_CHECK(hipMemcpy(dummy, zs_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, zs, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in zs: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, hh_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, hh, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in hh: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, uu_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, uu, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in uu: %f\n", maxdiffer);
	}

	CUDA_CHECK(hipMemcpy(dummy, vv_g, nx*ny * sizeof(float), hipMemcpyDeviceToHost));
	maxdiffer = maxdiff(nx*ny, vv, dummy);
	if (maxdiffer > maxerr)
	{
		printf("High error in vv: %f\n", maxdiffer);
	}


}




void LeftFlowBnd(Param XParam, std::vector<SLTS> leftWLbnd)
{
	//
	int nx = XParam.nx;
	int ny = XParam.ny;
	if (XParam.left == 1 && !leftWLbnd.empty())
	{
		int SLstepinbnd = 1;

		

		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = leftWLbnd[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = leftWLbnd[SLstepinbnd].time - XParam.totaltime;
		}

		

		dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((ny*1.0f) / blockDim.x), 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			double itime = SLstepinbnd - 1.0 + (XParam.totaltime - leftWLbnd[SLstepinbnd - 1].time) / (leftWLbnd[SLstepinbnd].time - leftWLbnd[SLstepinbnd - 1].time);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				leftdirichletD << <gridDim, blockDim, 0 >> > (nx, ny, (int)leftWLbnd[0].wlevs.size(), XParam.g, itime, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				leftdirichlet << <gridDim, blockDim, 0 >> > (nx, ny, (int)leftWLbnd[0].wlevs.size(), (float)XParam.g, (float)itime, zs_g, zb_g, hh_g, uu_g, vv_g);
			}
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<double> zsbndleft;
			for (int n = 0; n < leftWLbnd[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndleft.push_back( interptime(leftWLbnd[SLstepinbnd].wlevs[n], leftWLbnd[SLstepinbnd - 1].wlevs[n], leftWLbnd[SLstepinbnd].time - leftWLbnd[SLstepinbnd - 1].time, XParam.totaltime - leftWLbnd[SLstepinbnd - 1].time));

			}
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				leftdirichletCPUD(nx, ny, XParam.g, zsbndleft, zs_d, zb_d, hh_d, uu_d, vv_d);
			}
			else
			{
				leftdirichletCPU(nx, ny, (float)XParam.g, zsbndleft, zs, zb, hh, uu, vv);
			}
			
		}
	}
	if (XParam.left == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
			dim3 gridDim(ceil((ny*1.0) / blockDim.x), 1, 1);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				noslipbndLeft << <gridDim, blockDim, 0 >> > (nx, ny, XParam.eps, zb_gd, zs_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				noslipbndLeft << <gridDim, blockDim, 0 >> > (nx, ny, (float)XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
			}
			
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			noslipbndLCPU(XParam);
		}
	}
	//else neumann bnd (is already built in the solver)
}

void RightFlowBnd(Param XParam, std::vector<SLTS> rightWLbnd)
{
	//
	int nx = XParam.nx;
	int ny = XParam.ny;
	if (XParam.right == 1 && !rightWLbnd.empty())
	{
		int SLstepinbnd = 1;

		



		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = rightWLbnd[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = rightWLbnd[SLstepinbnd].time - XParam.totaltime;
		}

		

		dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((ny*1.0f) / blockDim.x), 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			double itime = SLstepinbnd - 1.0 + (XParam.totaltime - rightWLbnd[SLstepinbnd - 1].time) / (rightWLbnd[SLstepinbnd].time - rightWLbnd[SLstepinbnd - 1].time);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				rightdirichletD << <gridDim, blockDim, 0 >> > (nx, ny, (int)rightWLbnd[0].wlevs.size(), XParam.g, itime, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				rightdirichlet << <gridDim, blockDim, 0 >> > (nx, ny, (int)rightWLbnd[0].wlevs.size(), (float)XParam.g, (float)itime, zs_g, zb_g, hh_g, uu_g, vv_g);
			}
			
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<double> zsbndright;
			for (int n = 0; n < rightWLbnd[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndright.push_back( interptime(rightWLbnd[SLstepinbnd].wlevs[n], rightWLbnd[SLstepinbnd - 1].wlevs[n], rightWLbnd[SLstepinbnd].time - rightWLbnd[SLstepinbnd - 1].time, XParam.totaltime - rightWLbnd[SLstepinbnd - 1].time));

			}
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				rightdirichletCPUD(nx, ny, XParam.g, zsbndright, zs_d, zb_d, hh_d, uu_d, vv_d);
			}
			else
			{
				rightdirichletCPU(nx, ny, (float)XParam.g, zsbndright, zs, zb, hh, uu, vv);
			}
		}
	}
	if (XParam.right == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
			dim3 gridDim(ceil((ny*1.0) / blockDim.x), 1, 1);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				noslipbndRight << <gridDim, blockDim, 0 >> > (nx, ny, XParam.eps, zb_gd, zs_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				noslipbndRight << <gridDim, blockDim, 0 >> > (nx, ny, (float)XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
			}
			
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			void noslipbndRCPU(Param XParam);
		}
	}
	//else neumann bnd (is already built in the algorithm)
}

void TopFlowBnd(Param XParam, std::vector<SLTS> topWLbnd)
{
	//
	int nx = XParam.nx;
	int ny = XParam.ny;
	if (XParam.top == 1 && !topWLbnd.empty())
	{
		int SLstepinbnd = 1;





		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = topWLbnd[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = topWLbnd[SLstepinbnd].time - XParam.totaltime;
		}


		dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((nx*1.0) / blockDim.x), 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			double itime = SLstepinbnd - 1.0 + (XParam.totaltime - topWLbnd[SLstepinbnd - 1].time) / (topWLbnd[SLstepinbnd].time - topWLbnd[SLstepinbnd - 1].time);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				topdirichletD << <gridDim, blockDim, 0 >> > (nx, ny, (int)topWLbnd[0].wlevs.size(), XParam.g, itime, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				topdirichlet << <gridDim, blockDim, 0 >> > (nx, ny, (int)topWLbnd[0].wlevs.size(), (float)XParam.g, (float)itime, zs_g, zb_g, hh_g, uu_g, vv_g);
			}
			
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<double> zsbndtop;
			for (int n = 0; n < topWLbnd[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndtop.push_back( interptime(topWLbnd[SLstepinbnd].wlevs[n], topWLbnd[SLstepinbnd - 1].wlevs[n], topWLbnd[SLstepinbnd].time - topWLbnd[SLstepinbnd - 1].time, XParam.totaltime - topWLbnd[SLstepinbnd - 1].time));

			}
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				topdirichletCPUD(nx, ny, XParam.g, zsbndtop, zs_d, zb_d, hh_d, uu_d, vv_d);
			}
			else
			{

				topdirichletCPU(nx, ny, (float)XParam.g, zsbndtop, zs, zb, hh, uu, vv);
			}
		}
	}
	if (XParam.top == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
			dim3 gridDim(ceil((nx*1.0) / blockDim.x), 1, 1);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				noslipbndTop << <gridDim, blockDim, 0 >> > (nx, ny, XParam.eps, zb_gd, zs_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				noslipbndTop << <gridDim, blockDim, 0 >> > (nx, ny, (float)XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
			}
			
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			void noslipbndTCPU(Param XParam);
		}
	}
	//else neumann bnd (is already built in the algorithm)
}

void BotFlowBnd(Param XParam, std::vector<SLTS> botWLbnd)
{
	//
	int nx = XParam.nx;
	int ny = XParam.ny;
	if (XParam.bot == 1 && !botWLbnd.empty())
	{
		int SLstepinbnd = 1;





		// Do this for all the corners
		//Needs limiter in case WLbnd is empty
		double difft = botWLbnd[SLstepinbnd].time - XParam.totaltime;

		while (difft < 0.0)
		{
			SLstepinbnd++;
			difft = botWLbnd[SLstepinbnd].time - XParam.totaltime;
		}

		

		dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((nx*1.0) / blockDim.x), 1, 1);
		if (XParam.GPUDEVICE >= 0)
		{
			//leftdirichlet(int nx, int ny, int nybnd, float g, float itime, float *zs, float *zb, float *hh, float *uu, float *vv)
			double itime = SLstepinbnd - 1.0 + (XParam.totaltime - botWLbnd[SLstepinbnd - 1].time) / (botWLbnd[SLstepinbnd].time - botWLbnd[SLstepinbnd - 1].time);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				botdirichletD << <gridDim, blockDim, 0 >> > (nx, ny, (int)botWLbnd[0].wlevs.size(), XParam.g, itime, zs_gd, zb_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				botdirichlet << <gridDim, blockDim, 0 >> > (nx, ny, (int)botWLbnd[0].wlevs.size(), (float)XParam.g, (float)itime, zs_g, zb_g, hh_g, uu_g, vv_g);
			}
			
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			std::vector<double> zsbndbot;
			for (int n = 0; n < botWLbnd[SLstepinbnd].wlevs.size(); n++)
			{
				zsbndbot.push_back( interptime(botWLbnd[SLstepinbnd].wlevs[n], botWLbnd[SLstepinbnd - 1].wlevs[n], botWLbnd[SLstepinbnd].time - botWLbnd[SLstepinbnd - 1].time, XParam.totaltime - botWLbnd[SLstepinbnd - 1].time));

			}
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				botdirichletCPUD(nx, ny, XParam.g, zsbndbot, zs_d, zb_d, hh_d, uu_d, vv_d);
			}
			else
			{
				botdirichletCPU(nx, ny, (float)XParam.g, zsbndbot, zs, zb, hh, uu, vv);
			}
		}
	}
	if (XParam.bot == 0)
	{
		if (XParam.GPUDEVICE >= 0)
		{
			//
			dim3 blockDim(16, 1, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
			dim3 gridDim(ceil((nx*1.0) / blockDim.x), 1, 1);
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				noslipbndBot << <gridDim, blockDim, 0 >> > (nx, ny, XParam.eps, zb_gd, zs_gd, hh_gd, uu_gd, vv_gd);
			}
			else
			{
				noslipbndBot << <gridDim, blockDim, 0 >> > (nx, ny, (float)XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
			}
			
			CUDA_CHECK(hipDeviceSynchronize());
		}
		else
		{
			void noslipbndBCPU(Param XParam);
		}
	}
	//else neumann bnd (is already built in the algorithm)
}

double FlowGPU(Param XParam, double nextoutputtime)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	const int num_streams = 2;

	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++) 
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}



	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

	
	dtmax = (float) (1.0 / epsilon);
	//float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim,0, streams[0] >> > (nx, ny, dtmax_g);
	//CUDA_CHECK(hipDeviceSynchronize());
	//update step 1

	

	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, hh_g, dhdx_g, dhdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, zs_g, dzsdx_g, dzsdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, uu_g, dudx_g, dudy_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, vv_g, dvdx_g, dvdy_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipStreamSynchronize(streams[0]));
	//normal cartesian case
	updateKurgX << <gridDim, blockDim, 0, streams[0] >> > (nx, ny, (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, hh_g, zs_g, uu_g, vv_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipStreamSynchronize(streams[1]));
	updateKurgY << <gridDim, blockDim, 0, streams[1] >> > (nx, ny, (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, hh_g, zs_g, uu_g, vv_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);

	CUDA_CHECK(hipDeviceSynchronize());
	


	//GPU Harris reduction #3. 8.3x reduction #0  Note #7 if a lot faster
	// This was successfully tested with a range of grid size
	//reducemax3 << <gridDimLine, blockDimLine, 64*sizeof(float) >> >(dtmax_g, arrmax_g, nx*ny)
	int s = nx*ny;
	int maxThreads = 256;
	int threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
	int blocks = (s + (threads * 2 - 1)) / (threads * 2);
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);
	dim3 blockDimLine(threads, 1, 1);
	dim3 gridDimLine(blocks, 1, 1);

	float mindtmaxB;

	reducemin3 << <gridDimLine, blockDimLine, smemSize >> > (dtmax_g, arrmax_g, nx*ny);
	CUDA_CHECK(hipDeviceSynchronize());

	

	s = gridDimLine.x;
	while (s > 1)//cpuFinalThreshold
	{
		threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
		blocks = (s + (threads * 2 - 1)) / (threads * 2);

		smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

		dim3 blockDimLineS(threads, 1, 1);
		dim3 gridDimLineS(blocks, 1, 1);

		CUDA_CHECK(hipMemcpy(dtmax_g, arrmax_g, s * sizeof(float), hipMemcpyDeviceToDevice));

		reducemin3 << <gridDimLineS, blockDimLineS, smemSize >> > (dtmax_g, arrmax_g, s);
		CUDA_CHECK(hipDeviceSynchronize());

		s = (s + (threads * 2 - 1)) / (threads * 2);
	}

	
	CUDA_CHECK(hipMemcpy(dummy, arrmax_g, 32*sizeof(float), hipMemcpyDeviceToHost));
	mindtmaxB = dummy[0];
	/*
	//32 seem safe here bu I wonder why it is not 1 for the largers arrays...
	
	for (int i = 0; i < 32; i++)
	{
		mindtmaxB = min(dummy[i], mindtmaxB);
		printf("dt=%f\n", dummy[i]);
		
	}
	*/
	

	//float diffdt = mindtmaxB - mindtmax;
	XParam.dt = mindtmaxB;
	if (ceil((nextoutputtime - XParam.totaltime) / XParam.dt)> 0.0)
	{
		XParam.dt = (nextoutputtime - XParam.totaltime) / ceil((nextoutputtime - XParam.totaltime) / XParam.dt);
	}
	//printf("dt=%f\n", XParam.dt);

	
	updateEV << <gridDim, blockDim, 0 >> > (nx, ny, (float)XParam.delta, (float)XParam.g, hh_g, uu_g, vv_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	


	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.dt*0.5f, (float)XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//corrector setp
	//update again
	

	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, hho_g, dhdx_g, dhdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, zso_g, dzsdx_g, dzsdy_g);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, uuo_g, dudx_g, dudy_g);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, (float)XParam.theta, (float)XParam.delta, vvo_g, dvdx_g, dvdy_g);
	
	CUDA_CHECK(hipDeviceSynchronize());


	
	updateKurgX << <gridDim, blockDim, 0, streams[0] >> > (nx, ny, (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdx_g, dhdx_g, dudx_g, dvdx_g, Fhu_g, Fqux_g, Fqvx_g, Su_g, dtmax_g);
	//CUDA_CHECK(hipDeviceSynchronize());


	updateKurgY << <gridDim, blockDim, 0, streams[1] >> > (nx, ny, (float)XParam.delta, (float)XParam.g, (float)XParam.eps, (float)XParam.CFL, hho_g, zso_g, uuo_g, vvo_g, dzsdy_g, dhdy_g, dudy_g, dvdy_g, Fhv_g, Fqvy_g, Fquy_g, Sv_g, dtmax_g);
	CUDA_CHECK(hipDeviceSynchronize());
	
	// no reduction of dtmax during the corrector step

	
	updateEV << <gridDim, blockDim, 0 >> > (nx, ny, (float)XParam.delta, (float)XParam.g, hho_g, uuo_g, vvo_g, Fhu_g, Fhv_g, Su_g, Sv_g, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g, dh_g, dhu_g, dhv_g);
	CUDA_CHECK(hipDeviceSynchronize());
	
	

	//
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, (float)XParam.dt, (float)XParam.eps, hh_g, zb_g, uu_g, vv_g, dh_g, dhu_g, dhv_g, zso_g, hho_g, uuo_g, vvo_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);
	cleanupGPU << <gridDim, blockDim, 0 >> >(nx, ny, hho_g, zso_g, uuo_g, vvo_g, hh_g, zs_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	//Bottom friction
	quadfriction << <gridDim, blockDim, 0 >> > (nx, ny, (float)XParam.dt, (float)XParam.eps, (float)XParam.cf, hh_g, uu_g, vv_g);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipStreamDestroy(streams[0]));
	CUDA_CHECK(hipStreamDestroy(streams[1]));

	// Impose no slip condition by default
	//noslipbndall << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
	//CUDA_CHECK(hipDeviceSynchronize());
	return XParam.dt;
}

double FlowGPUSpherical(Param XParam, double nextoutputtime)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	const int num_streams = 2;

	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}



	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0) / blockDim.x), ceil((ny*1.0) / blockDim.y), 1);


	dtmax = (float)(1.0 / epsilon);
	//float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0, streams[0] >> > (nx, ny, dtmax_gd);
	//CUDA_CHECK(hipDeviceSynchronize());
	//update step 1

	


	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, XParam.theta, XParam.delta, hh_gd, dhdx_gd, dhdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, XParam.theta, XParam.delta, zs_gd, dzsdx_gd, dzsdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());


	

	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, XParam.theta, XParam.delta, uu_gd, dudx_gd, dudy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	

	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, XParam.theta, XParam.delta, vv_gd, dvdx_gd, dvdy_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipStreamSynchronize(streams[0]));
	//Spherical
	{
		//Spherical coordinates 
		updateKurgXSPH << <gridDim, blockDim, 0, streams[0] >> > (nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, XParam.yo, XParam.Radius, hh_gd, zs_gd, uu_gd, vv_gd, dzsdx_gd, dhdx_gd, dudx_gd, dvdx_gd, Fhu_gd, Fqux_gd, Fqvx_gd, Su_gd, dtmax_gd);

		updateKurgYSPH << <gridDim, blockDim, 0, streams[1] >> > (nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, XParam.yo, XParam.Radius, hh_gd, zs_gd, uu_gd, vv_gd, dzsdy_gd, dhdy_gd, dudy_gd, dvdy_gd, Fhv_gd, Fqvy_gd, Fquy_gd, Sv_gd, dtmax_gd);

		CUDA_CHECK(hipDeviceSynchronize());

	}

	/////////////////////////////////////////////////////
	// Reduction of dtmax
	/////////////////////////////////////////////////////

	// copy from GPU and do the reduction on the CPU  ///LAME!
	

	//GPU Harris reduction #3. 8.3x reduction #0  Note #7 if a lot faster
	// This was successfully tested with a range of grid size
	//reducemax3 << <gridDimLine, blockDimLine, 64*sizeof(float) >> >(dtmax_g, arrmax_g, nx*ny)
	int s = nx*ny;
	int maxThreads = 256;
	int threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
	int blocks = (s + (threads * 2 - 1)) / (threads * 2);
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);
	dim3 blockDimLine(threads, 1, 1);
	dim3 gridDimLine(blocks, 1, 1);

	double mindtmaxB;

	reducemin3 << <gridDimLine, blockDimLine, smemSize >> > (dtmax_gd, arrmax_gd, nx*ny);
	CUDA_CHECK(hipDeviceSynchronize());



	s = gridDimLine.x;
	while (s > 1)//cpuFinalThreshold
	{
		threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
		blocks = (s + (threads * 2 - 1)) / (threads * 2);

		smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

		dim3 blockDimLineS(threads, 1, 1);
		dim3 gridDimLineS(blocks, 1, 1);

		CUDA_CHECK(hipMemcpy(dtmax_gd, arrmax_gd, s * sizeof(double), hipMemcpyDeviceToDevice));

		reducemin3 << <gridDimLineS, blockDimLineS, smemSize >> > (dtmax_gd, arrmax_gd, s);
		CUDA_CHECK(hipDeviceSynchronize());

		s = (s + (threads * 2 - 1)) / (threads * 2);
	}


	CUDA_CHECK(hipMemcpy(dummy_d, arrmax_gd, 32 * sizeof(double), hipMemcpyDeviceToHost));
	mindtmaxB = dummy_d[0];
	/*
	//32 seem safe here bu I wonder why it is not 1 for the largers arrays...

	for (int i = 0; i < 32; i++)
	{
	mindtmaxB = min(dummy[i], mindtmaxB);
	printf("dt=%f\n", dummy[i]);

	}
	*/


	//float diffdt = mindtmaxB - mindtmax;
	XParam.dt = mindtmaxB;
	if (ceil((nextoutputtime - XParam.totaltime) / XParam.dt)> 0.0)
	{
		XParam.dt = (nextoutputtime - XParam.totaltime) / ceil((nextoutputtime - XParam.totaltime) / XParam.dt);
	}
	//printf("dt=%f\n", XParam.dt);

	//spherical
	{
		//if spherical corrdinate use this kernel with the right corrections
		updateEVSPH << <gridDim, blockDim, 0 >> > (nx, ny, XParam.delta, XParam.g, XParam.yo, XParam.Radius, hh_gd, uu_gd, vv_gd, Fhu_gd, Fhv_gd, Su_gd, Sv_gd, Fqux_gd, Fquy_gd, Fqvx_gd, Fqvy_gd, dh_gd, dhu_gd, dhv_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}


	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt*0.5, XParam.eps, hh_gd, zb_gd, uu_gd, vv_gd, dh_gd, dhu_gd, dhv_gd, zso_gd, hho_gd, uuo_gd, vvo_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//corrector setp
	//update again
	// calculate gradients
	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdy_g);

	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, XParam.theta, XParam.delta, hho_gd, dhdx_gd, dhdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdy_g);

	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, XParam.theta, XParam.delta, zso_gd, dzsdx_gd, dzsdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudy_g);

	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, XParam.theta, XParam.delta, uuo_gd, dudx_gd, dudy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdy_g);

	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, XParam.theta, XParam.delta, vvo_gd, dvdx_gd, dvdy_gd);

	CUDA_CHECK(hipDeviceSynchronize());


	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	//CUDA_CHECK(hipDeviceSynchronize());

	
	{
		//Spherical coordinates 
		updateKurgXSPH << <gridDim, blockDim, 0, streams[0] >> > (nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, XParam.yo, XParam.Radius, hho_gd, zso_gd, uuo_gd, vvo_gd, dzsdx_gd, dhdx_gd, dudx_gd, dvdx_gd, Fhu_gd, Fqux_gd, Fqvx_gd, Su_gd, dtmax_gd);

		updateKurgYSPH << <gridDim, blockDim, 0, streams[1] >> > (nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, XParam.yo, XParam.Radius, hho_gd, zso_gd, uuo_gd, vvo_gd, dzsdy_gd, dhdy_gd, dudy_gd, dvdy_gd, Fhv_gd, Fqvy_gd, Fquy_gd, Sv_gd, dtmax_gd);

		CUDA_CHECK(hipDeviceSynchronize());

	}
	// no reduction of dtmax during the corrector step

	
	{
		//if spherical corrdinate use this kernel with the right corrections
		updateEVSPH << <gridDim, blockDim, 0 >> > (nx, ny, XParam.delta, XParam.g, XParam.yo, XParam.Radius, hho_gd, uuo_gd, vvo_gd, Fhu_gd, Fhv_gd, Su_gd, Sv_gd, Fqux_gd, Fquy_gd, Fqvx_gd, Fqvy_gd, dh_gd, dhu_gd, dhv_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}

	//
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt, XParam.eps, hh_gd, zb_gd, uu_gd, vv_gd, dh_gd, dhu_gd, dhv_gd, zso_gd, hho_gd, uuo_gd, vvo_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);
	cleanupGPU << <gridDim, blockDim, 0 >> >(nx, ny, hho_gd, zso_gd, uuo_gd, vvo_gd, hh_gd, zs_gd, uu_gd, vv_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//Bottom friction
	quadfriction << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, XParam.cf, hh_gd, uu_gd, vv_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipStreamDestroy(streams[0]));
	CUDA_CHECK(hipStreamDestroy(streams[1]));

	// Impose no slip condition by default
	//noslipbndall << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
	//CUDA_CHECK(hipDeviceSynchronize());
	return XParam.dt;
}


double FlowGPUDouble(Param XParam, double nextoutputtime)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	const int num_streams = 2;

	hipStream_t streams[num_streams];
	for (int i = 0; i < num_streams; i++)
	{
		CUDA_CHECK(hipStreamCreate(&streams[i]));
	}



	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);


	dtmax = (float)(1.0 / epsilon);
	//float dtmaxtmp = dtmax;

	resetdtmax << <gridDim, blockDim, 0, streams[0] >> > (nx, ny, dtmax_gd);
	//CUDA_CHECK(hipDeviceSynchronize());
	//update step 1




	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, XParam.theta, XParam.delta, hh_gd, dhdx_gd, dhdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, XParam.theta, XParam.delta, zs_gd, dzsdx_gd, dzsdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());




	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, XParam.theta, XParam.delta, uu_gd, dudx_gd, dudy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());



	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, XParam.theta, XParam.delta, vv_gd, dvdx_gd, dvdy_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//CUDA_CHECK(hipStreamSynchronize(streams[0]));
	
	
		
	updateKurgXD << <gridDim, blockDim, 0, streams[0] >> > (nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL,  hh_gd, zs_gd, uu_gd, vv_gd, dzsdx_gd, dhdx_gd, dudx_gd, dvdx_gd, Fhu_gd, Fqux_gd, Fqvx_gd, Su_gd, dtmax_gd);

	updateKurgYD << <gridDim, blockDim, 0, streams[1] >> > (nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL,  hh_gd, zs_gd, uu_gd, vv_gd, dzsdy_gd, dhdy_gd, dudy_gd, dvdy_gd, Fhv_gd, Fqvy_gd, Fquy_gd, Sv_gd, dtmax_gd);

	CUDA_CHECK(hipDeviceSynchronize());

	

	/////////////////////////////////////////////////////
	// Reduction of dtmax
	/////////////////////////////////////////////////////

	// copy from GPU and do the reduction on the CPU  ///LAME!


	//GPU Harris reduction #3. 8.3x reduction #0  Note #7 if a lot faster
	// This was successfully tested with a range of grid size
	//reducemax3 << <gridDimLine, blockDimLine, 64*sizeof(float) >> >(dtmax_g, arrmax_g, nx*ny)
	int s = nx*ny;
	int maxThreads = 256;
	int threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
	int blocks = (s + (threads * 2 - 1)) / (threads * 2);
	int smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);
	dim3 blockDimLine(threads, 1, 1);
	dim3 gridDimLine(blocks, 1, 1);

	double mindtmaxB;

	reducemin3 << <gridDimLine, blockDimLine, smemSize >> > (dtmax_gd, arrmax_gd, nx*ny);
	CUDA_CHECK(hipDeviceSynchronize());



	s = gridDimLine.x;
	while (s > 1)//cpuFinalThreshold
	{
		threads = (s < maxThreads * 2) ? nextPow2((s + 1) / 2) : maxThreads;
		blocks = (s + (threads * 2 - 1)) / (threads * 2);

		smemSize = (threads <= 32) ? 2 * threads * sizeof(double) : threads * sizeof(double);

		dim3 blockDimLineS(threads, 1, 1);
		dim3 gridDimLineS(blocks, 1, 1);

		CUDA_CHECK(hipMemcpy(dtmax_gd, arrmax_gd, s * sizeof(float), hipMemcpyDeviceToDevice));

		reducemin3 << <gridDimLineS, blockDimLineS, smemSize >> > (dtmax_gd, arrmax_gd, s);
		CUDA_CHECK(hipDeviceSynchronize());

		s = (s + (threads * 2 - 1)) / (threads * 2);
	}


	CUDA_CHECK(hipMemcpy(dummy_d, arrmax_gd, 32 * sizeof(float), hipMemcpyDeviceToHost));
	mindtmaxB = dummy_d[0];
	/*
	//32 seem safe here bu I wonder why it is not 1 for the largers arrays...

	for (int i = 0; i < 32; i++)
	{
	mindtmaxB = min(dummy[i], mindtmaxB);
	printf("dt=%f\n", dummy[i]);

	}
	*/


	//float diffdt = mindtmaxB - mindtmax;
	XParam.dt = mindtmaxB;
	if (ceil((nextoutputtime - XParam.totaltime) / XParam.dt)> 0.0)
	{
		XParam.dt = (nextoutputtime - XParam.totaltime) / ceil((nextoutputtime - XParam.totaltime) / XParam.dt);
	}
	//printf("dt=%f\n", XParam.dt);

	
		//if spherical corrdinate use this kernel with the right corrections
	updateEVD << <gridDim, blockDim, 0 >> > (nx, ny, XParam.delta, XParam.g, hh_gd, uu_gd, vv_gd, Fhu_gd, Fhv_gd, Su_gd, Sv_gd, Fqux_gd, Fquy_gd, Fqvx_gd, Fqvy_gd, dh_gd, dhu_gd, dhv_gd);
	CUDA_CHECK(hipDeviceSynchronize());
	


	//predictor (advance 1/2 dt)
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt*0.5, XParam.eps, hh_gd, zb_gd, uu_gd, vv_gd, dh_gd, dhu_gd, dhv_gd, zso_gd, hho_gd, uuo_gd, vvo_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//corrector setp
	//update again
	// calculate gradients
	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, hho_g, dhdy_g);

	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, XParam.theta, XParam.delta, hho_gd, dhdx_gd, dhdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, zso_g, dzsdy_g);

	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, XParam.theta, XParam.delta, zso_gd, dzsdx_gd, dzsdy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, uuo_g, dudy_g);

	gradientGPUXY << <gridDim, blockDim, 0, streams[0] >> >(nx, ny, XParam.theta, XParam.delta, uuo_gd, dudx_gd, dudy_gd);
	//CUDA_CHECK(hipDeviceSynchronize());

	//gradientGPUX << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdx_g);
	//gradientGPUY << <gridDim, blockDim, 0 >> >(nx, ny, XParam.theta, XParam.delta, vvo_g, dvdy_g);

	gradientGPUXY << <gridDim, blockDim, 0, streams[1] >> >(nx, ny, XParam.theta, XParam.delta, vvo_gd, dvdx_gd, dvdy_gd);

	CUDA_CHECK(hipDeviceSynchronize());


	// Test whether it is better to have one here or later (are the instuctions overlap if occupancy and meme acess is available?)
	//CUDA_CHECK(hipDeviceSynchronize());


	
	updateKurgXD << <gridDim, blockDim, 0, streams[0] >> > (nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, hho_gd, zso_gd, uuo_gd, vvo_gd, dzsdx_gd, dhdx_gd, dudx_gd, dvdx_gd, Fhu_gd, Fqux_gd, Fqvx_gd, Su_gd, dtmax_gd);

	updateKurgYD << <gridDim, blockDim, 0, streams[1] >> > (nx, ny, XParam.delta, XParam.g, XParam.eps, XParam.CFL, hho_gd, zso_gd, uuo_gd, vvo_gd, dzsdy_gd, dhdy_gd, dudy_gd, dvdy_gd, Fhv_gd, Fqvy_gd, Fquy_gd, Sv_gd, dtmax_gd);

	CUDA_CHECK(hipDeviceSynchronize());

	
	// no reduction of dtmax during the corrector step


	
	
	updateEVD << <gridDim, blockDim, 0 >> > (nx, ny, XParam.delta, XParam.g,  hho_gd, uuo_gd, vvo_gd, Fhu_gd, Fhv_gd, Su_gd, Sv_gd, Fqux_gd, Fquy_gd, Fqvx_gd, Fqvy_gd, dh_gd, dhu_gd, dhv_gd);
	CUDA_CHECK(hipDeviceSynchronize());
	

	//
	Advkernel << <gridDim, blockDim, 0 >> >(nx, ny, XParam.dt, XParam.eps, hh_gd, zb_gd, uu_gd, vv_gd, dh_gd, dhu_gd, dhv_gd, zso_gd, hho_gd, uuo_gd, vvo_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//cleanup(nx, ny, hho, zso, uuo, vvo, hh, zs, uu, vv);
	cleanupGPU << <gridDim, blockDim, 0 >> >(nx, ny, hho_gd, zso_gd, uuo_gd, vvo_gd, hh_gd, zs_gd, uu_gd, vv_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	//Bottom friction
	quadfriction << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, XParam.cf, hh_gd, uu_gd, vv_gd);
	CUDA_CHECK(hipDeviceSynchronize());

	CUDA_CHECK(hipStreamDestroy(streams[0]));
	CUDA_CHECK(hipStreamDestroy(streams[1]));

	// Impose no slip condition by default
	//noslipbndall << <gridDim, blockDim, 0 >> > (nx, ny, XParam.dt, XParam.eps, zb_g, zs_g, hh_g, uu_g, vv_g);
	//CUDA_CHECK(hipDeviceSynchronize());
	return XParam.dt;
}


void meanmaxvarGPU(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, uumean_g, uu_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, vvmean_g, vv_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmean_g, hh_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmean_g, zs_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmax_g, zs_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmax_g, hh_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outuumax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, uumax_g, uu_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, vvmax_g, vv_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}

}


void meanmaxvarGPUD(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0) / blockDim.x), ceil((ny*1.0) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, uumean_gd, uu_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, vvmean_gd, vv_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmean_gd, hh_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmean == 1)
	{
		addavg_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmean_gd, zs_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outzsmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmax_gd, zs_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outhhmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmax_gd, hh_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outuumax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, uumax_gd, uu_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	if (XParam.outvvmax == 1)
	{
		max_var << <gridDim, blockDim, 0 >> >(nx, ny, vvmax_gd, vv_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}

}


void DivmeanvarGPU(Param XParam, float nstep)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, uumean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outvvmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, vvmean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outhhmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, hhmean_g);
		CUDA_CHECK(hipDeviceSynchronize());

		
	}
	if (XParam.outzsmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, zsmean_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
	
	

}


void DivmeanvarGPUD(Param XParam, double nstep)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0) / blockDim.x), ceil((ny*1.0) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, uumean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, vvmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, hhmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmean == 1)
	{
		divavg_var << <gridDim, blockDim, 0 >> >(nx, ny, nstep, zsmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}



}

void ResetmeanvarGPU(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, uumean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny,  vvmean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmean_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmean_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
}




void ResetmeanvarGPUD(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0) / blockDim.x), ceil((ny*1.0) / blockDim.y), 1);
	if (XParam.outuumean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, uumean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, vvmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmean == 1)
	{
		resetavg_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmean_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
}
void ResetmaxvarGPU(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.outuumax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(nx, ny, uumax_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(nx, ny, vvmax_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmax_g);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmax_g);
		CUDA_CHECK(hipDeviceSynchronize());
	}
}
void ResetmaxvarGPUD(Param XParam)
{
	int nx = XParam.nx;
	int ny = XParam.ny;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);
	if (XParam.outuumax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(nx, ny, uumax_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outvvmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(nx, ny, vvmax_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outhhmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(nx, ny, hhmax_gd);
		CUDA_CHECK(hipDeviceSynchronize());


	}
	if (XParam.outzsmax == 1)
	{
		resetmax_var << <gridDim, blockDim, 0 >> >(nx, ny, zsmax_gd);
		CUDA_CHECK(hipDeviceSynchronize());
	}
}

// Main loop that actually runs the model
void mainloopGPU(Param XParam, std::vector<SLTS> leftWLbnd, std::vector<SLTS> rightWLbnd, std::vector<SLTS> topWLbnd, std::vector<SLTS> botWLbnd)
{
	double nextoutputtime = XParam.outputtimestep;
	int nstep = 0;
	int nTSsteps = 0;

	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
	dim3 gridDim(ceil((XParam.nx*1.0) / blockDim.x), ceil((XParam.ny*1.0) / blockDim.y), 1);
	


	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}
	// Reset GPU mean and max arrays
	if (XParam.spherical == 1 || XParam.doubleprecision == 1)
	{
		ResetmeanvarGPUD(XParam);
		ResetmaxvarGPUD(XParam);
	}
	else
	{
		ResetmeanvarGPU(XParam);
		ResetmaxvarGPU(XParam);
	}

	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam, leftWLbnd);
		RightFlowBnd(XParam, rightWLbnd);
		TopFlowBnd(XParam, topWLbnd);
		BotFlowBnd(XParam, botWLbnd);

		// Run the model step
		if (XParam.spherical == 1)
		{
			XParam.dt = FlowGPUSpherical(XParam, nextoutputtime);
		}
		else
		{
			if(XParam.doubleprecision==1)
			{
				XParam.dt = FlowGPUDouble(XParam, nextoutputtime);
			}
			else
			{
				XParam.dt = FlowGPU(XParam, nextoutputtime);
			}
			
		}
		
		
		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;
		
		// Do Sum & Max variables Here
		if (XParam.spherical == 1 || XParam.doubleprecision == 1)
		{
			meanmaxvarGPUD(XParam);
		}
		else
		{
			meanmaxvarGPU(XParam);
		}
		


		//Check for TSoutput
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = 0.0;// a bit useless this
				stepread.hh = 0.0;
				stepread.uu = 0.0;
				stepread.vv = 0.0;
				zsAllout[o].push_back(stepread);

				if (XParam.spherical == 1 || XParam.doubleprecision == 1)
				{
					storeTSout << <gridDim, blockDim, 0 >> > (XParam.nx, XParam.ny, (int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, zs_gd, hh_gd, uu_gd, vv_gd, TSstore_gd);
				}
				else
				{
					storeTSout << <gridDim, blockDim, 0 >> > (XParam.nx, XParam.ny, (int)XParam.TSnodesout.size(), o, nTSsteps, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, zs_g, hh_g, uu_g, vv_g, TSstore_g);
				}
				
				CUDA_CHECK(hipDeviceSynchronize());
			}
			nTSsteps++;
			
			if ((nTSsteps+1)*XParam.TSnodesout.size() * 4 > 2048 || XParam.endtime-XParam.totaltime <= XParam.dt*0.00001f)
			{
				//Flush
				if (XParam.spherical == 1 || XParam.doubleprecision == 1)
				{
					CUDA_CHECK(hipMemcpy(TSstore_d, TSstore_gd, 2048 * sizeof(double), hipMemcpyDeviceToHost));
					for (int o = 0; o < XParam.TSnodesout.size(); o++)
					{
						fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
						for (int n = 0; n < nTSsteps; n++)
						{
							//


							fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, TSstore_d[1 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[0 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[2 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore_d[3 + o * 4 + n*XParam.TSnodesout.size() * 4]);


						}
						fclose(fsSLTS);
						//reset zsout
						zsAllout[o].clear();
					}
					nTSsteps = 0;
				}
				else
				{

					CUDA_CHECK(hipMemcpy(TSstore, TSstore_g, 2048 * sizeof(float), hipMemcpyDeviceToHost));
					for (int o = 0; o < XParam.TSnodesout.size(); o++)
					{
						fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
						for (int n = 0; n < nTSsteps; n++)
						{
							//


							fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, TSstore[1 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[0 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[2 + o * 4 + n*XParam.TSnodesout.size() * 4], TSstore[3 + o * 4 + n*XParam.TSnodesout.size() * 4]);


						}
						fclose(fsSLTS);
						//reset zsout
						zsAllout[o].clear();
					}
					nTSsteps = 0;
				}

				

			}
			

		}

		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			if (XParam.spherical == 1 || XParam.doubleprecision == 1)
			{
				DivmeanvarGPUD(XParam, nstep);

				if (XParam.outvort == 1)
				{
					CalcVorticity << <gridDim, blockDim, 0 >> > (XParam.nx, XParam.ny, vort_gd, dvdx_gd, dudy_gd);
					CUDA_CHECK(hipDeviceSynchronize());
				}

				if (!XParam.outvars.empty())
				{
					writenctimestep(XParam.outfile, XParam.totaltime);

					for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
					{
						if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
						{
							if (XParam.GPUDEVICE >= 0)
							{
								//Should be async
								CUDA_CHECK(hipMemcpy(OutputVarMapCPUD[XParam.outvars[ivar]], OutputVarMapGPUD[XParam.outvars[ivar]], OutputVarMaplen[XParam.outvars[ivar]] * sizeof(double), hipMemcpyDeviceToHost));

							}
							//Create definition for each variable and store it
							writencvarstepD(XParam.outfile, XParam.smallnc, XParam.scalefactor, XParam.addoffset, XParam.outvars[ivar], OutputVarMapCPUD[XParam.outvars[ivar]]);
						}
					}
				}
			}
			else
			{

				// Avg var sum here
				DivmeanvarGPU(XParam, nstep*1.0f);

				if (XParam.outvort == 1)
				{
					CalcVorticity << <gridDim, blockDim, 0 >> > (XParam.nx, XParam.ny, vort_g, dvdx_g, dudy_g);
					CUDA_CHECK(hipDeviceSynchronize());
				}

				if (!XParam.outvars.empty())
				{
					writenctimestep(XParam.outfile, XParam.totaltime);

					for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
					{
						if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
						{
							if (XParam.GPUDEVICE >= 0)
							{
								//Should be async
								CUDA_CHECK(hipMemcpy(OutputVarMapCPU[XParam.outvars[ivar]], OutputVarMapGPU[XParam.outvars[ivar]], OutputVarMaplen[XParam.outvars[ivar]] * sizeof(float), hipMemcpyDeviceToHost));

							}
							//Create definition for each variable and store it
							writencvarstep(XParam.outfile, XParam.smallnc, XParam.scalefactor, XParam.addoffset, XParam.outvars[ivar], OutputVarMapCPU[XParam.outvars[ivar]]);
						}
					}
				}
			}
			nextoutputtime = min(nextoutputtime + XParam.outputtimestep,XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//.Reset Avg Variables
			if (XParam.spherical == 1 || XParam.doubleprecision == 1)
			{
				ResetmeanvarGPUD(XParam);
				if (XParam.resetmax == 1)
				{
					ResetmaxvarGPUD(XParam);
				}
			}
			else
			{
				ResetmeanvarGPU(XParam);
				if (XParam.resetmax == 1)
				{
					ResetmaxvarGPU(XParam);
				}
			}
			


			//

			// Reset nstep
			nstep = 0;
		} // End of output part

	} //Main while loop
}




void mainloopCPU(Param XParam, std::vector<SLTS> leftWLbnd, std::vector<SLTS> rightWLbnd, std::vector<SLTS> topWLbnd, std::vector<SLTS> botWLbnd)
{
	double nextoutputtime = XParam.outputtimestep;
	int nstep = 0;

	int nTSstep = 0;


	std::vector<Pointout> zsout;

	std::vector< std::vector< Pointout > > zsAllout;

	Pointout stepread;

	FILE * fsSLTS;

	for (int o = 0; o < XParam.TSoutfile.size(); o++)
	{
		//Overwrite existing files
		fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "w");
		fprintf(fsSLTS, "# x=%f\ty=%f\ti=%d\tj=%d\t%s\n", XParam.TSnodesout[o].x, XParam.TSnodesout[o].y, XParam.TSnodesout[o].i, XParam.TSnodesout[o].j, XParam.TSoutfile[o].c_str());
		fclose(fsSLTS);

		// Add empty row for each output point
		zsAllout.push_back(std::vector<Pointout>());
	}

	while (XParam.totaltime < XParam.endtime)
	{
		// Bnd stuff here
		LeftFlowBnd(XParam, leftWLbnd);
		RightFlowBnd(XParam, rightWLbnd);
		TopFlowBnd(XParam, topWLbnd);
		BotFlowBnd(XParam, botWLbnd);


		// Run the model step
		if (XParam.spherical == 1)
		{
			XParam.dt = FlowCPUSpherical(XParam, nextoutputtime);
		}
		else
		{
			if (XParam.doubleprecision==1)
			{
				XParam.dt = FlowCPUDouble(XParam, nextoutputtime);
			}
			else
			{
				XParam.dt = FlowCPU(XParam, nextoutputtime);
			}
		}

		//Time keeping
		XParam.totaltime = XParam.totaltime + XParam.dt;
		nstep++;

		// Do Sum & Max variables Here
		if (XParam.doubleprecision == 1 || XParam.spherical == 1)
		{
			AddmeanCPUD(XParam);
			maxallCPUD(XParam);
		}
		else
		{
			AddmeanCPU(XParam);
			maxallCPU(XParam);
		}
		//Check for TSoutput
		if (XParam.TSnodesout.size() > 0)
		{
			for (int o = 0; o < XParam.TSnodesout.size(); o++)
			{
				//
				stepread.time = XParam.totaltime;
				stepread.zs = zs[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.hh = hh[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.uu = uu[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				stepread.vv = vv[XParam.TSnodesout[o].i + XParam.TSnodesout[o].j*XParam.nx];
				zsAllout[o].push_back(stepread);

			}
			nTSstep++;

		}
		// CHeck for grid output
		if (nextoutputtime - XParam.totaltime <= XParam.dt*0.00001f  && XParam.outputtimestep > 0)
		{
			// Avg var sum here

			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				DivmeanCPUD(XParam, (double)nstep);
				if (XParam.outvort == 1)
				{
					CalcVortD(XParam);
				}
			}
			else
			{
				DivmeanCPU(XParam, (float)nstep);
				if (XParam.outvort == 1)
				{
					CalcVort(XParam);
				}
			}
			
			// Check for and calculate Vorticity if required
			

			if (!XParam.outvars.empty())
			{
				writenctimestep(XParam.outfile, XParam.totaltime);

				for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
				{
					if (OutputVarMaplen[XParam.outvars[ivar]] > 0)
					{
						
						//write output step for each variable 
						if (XParam.doubleprecision == 1 || XParam.spherical == 1)
						{
							writencvarstepD(XParam.outfile, XParam.smallnc, XParam.scalefactor, XParam.addoffset, XParam.outvars[ivar], OutputVarMapCPUD[XParam.outvars[ivar]]);
						}
						else
						{
							writencvarstep(XParam.outfile, XParam.smallnc, XParam.scalefactor, XParam.addoffset, XParam.outvars[ivar], OutputVarMapCPU[XParam.outvars[ivar]]);
						}
						
					}
				}
			}
			nextoutputtime = min(nextoutputtime + XParam.outputtimestep, XParam.endtime);

			printf("Writing output, totaltime:%f s, Mean dt=%f\n", XParam.totaltime, XParam.outputtimestep / nstep);
			write_text_to_log_file("Writing outputs, totaltime: " + std::to_string(XParam.totaltime) + ", Mean dt= " + std::to_string(XParam.outputtimestep / nstep));

			//.Reset Avg Variables

			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				ResetmeanCPUD(XParam);
			}
			else
			{
				ResetmeanCPU(XParam);
			}
			

			//
			if (!XParam.TSoutfile.empty())
			{
				for (int o = 0; o < XParam.TSoutfile.size(); o++)
				{
					//Overwrite existing files
					fsSLTS = fopen(XParam.TSoutfile[o].c_str(), "a");
					for (int n = 0; n < zsAllout[o].size(); n++)
					{
						fprintf(fsSLTS, "%f\t%.4f\t%.4f\t%.4f\t%.4f\n", zsAllout[o][n].time, zsAllout[o][n].zs, zsAllout[o][n].hh, zsAllout[o][n].uu, zsAllout[o][n].vv);
					}
					fclose(fsSLTS);
					//reset zsout
					zsAllout[o].clear();
					//zsAllout.push_back(std::vector<SLBnd>());
				}
			}
			// Reset nstep
			nstep = 0;
		}

		

	}
}




int main(int argc, char **argv)
{

	
	//Model starts Here//
	Param XParam;
	//The main function setups all the init of the model and then calls the mainloop to actually run the model


	//First part reads the inputs to the model 
	//then allocate memory on GPU and CPU
	//Then prepare and initialise memory and arrays on CPU and GPU
	// Prepare output file
	// Run main loop
	// Clean up and close


	// Start timer to keep track of time 
	XParam.startcputime = clock();



	// Reset the log file 
	FILE * flog;
	flog = fopen("BG_log.txt", "w"); //Find better name
	fclose(flog);

	//Logfile header
	time_t rawtime;
	struct tm * timeinfo;
	char buffer[80];

	time(&rawtime);
	timeinfo = localtime(&rawtime);

	strftime(buffer, 80, "%d-%m-%Y %H:%M:%S", timeinfo);
	std::string strtimenow(buffer);
	write_text_to_log_file("#################################");
	write_text_to_log_file("Basilisk-like Cartesian GPU v0.0");
	write_text_to_log_file("#################################");
	write_text_to_log_file("model started at " + strtimenow);


	//////////////////////////////////////////////////////
	/////             Read Operational file          /////
	//////////////////////////////////////////////////////


	std::ifstream fs("BG_param.txt");

	if (fs.fail()) {
		std::cerr << "BG_param.txt file could not be opened" << std::endl;
		write_text_to_log_file("ERROR: BG_param.txt file could not be opened...use this log file to create a file named BG_param.txt");
		SaveParamtolog(XParam);

		exit(1);
		
	}
	else
	{
		// Read and interpret each line of the BG_param.txt
		std::string line;
		while (std::getline(fs, line))
		{
			
			//Get param or skip empty lines
			if (!line.empty() && line.substr(0, 1).compare("#") != 0)
			{
				XParam = readparamstr(line, XParam);
				//std::cout << line << std::endl;
			}

		}
		fs.close();

		
	}





	std::string bathyext;
	
	//read bathy and perform sanity check
		
	if (!XParam.Bathymetryfile.empty())
	{
		printf("bathy: %s\n", XParam.Bathymetryfile.c_str());

		write_text_to_log_file("bathy: " + XParam.Bathymetryfile);

		std::vector<std::string> extvec = split(XParam.Bathymetryfile, '.');

		std::vector<std::string> nameelements;
		//by default we expect tab delimitation
		nameelements = split(extvec.back(), '?');
		if (nameelements.size() > 1)
		{
			//variable name for bathy is not given so it is assumed to be zb
			bathyext = nameelements[0];
		}
		else
		{
			bathyext = extvec.back();
		}

		
		write_text_to_log_file("bathy extension: " + bathyext);
		if (bathyext.compare("md") == 0)
		{
			write_text_to_log_file("Reading 'md' file");
			readbathyHead(XParam.Bathymetryfile, XParam.nx, XParam.ny, XParam.dx, XParam.grdalpha);
			
		}
		if (bathyext.compare("nc") == 0)
		{
			write_text_to_log_file("Reading bathy netcdf file");
			readgridncsize(XParam.Bathymetryfile, XParam.nx, XParam.ny, XParam.dx);
			write_text_to_log_file("For nc of bathy file please specify grdalpha in the BG_param.txt (default 0)");
			

		}
		if (bathyext.compare("dep") == 0 || bathyext.compare("bot") == 0)
		{
			//XBeach style file
			write_text_to_log_file("Reading " + bathyext + " file");
			write_text_to_log_file("For this type of bathy file please specify nx, ny, dx, xo, yo and grdalpha in the XBG_param.txt");
		}
		if (bathyext.compare("asc") == 0)
		{
			//
			write_text_to_log_file("Reading bathy asc file");
			readbathyASCHead(XParam.Bathymetryfile, XParam.nx, XParam.ny, XParam.dx, XParam.xo, XParam.yo, XParam.grdalpha);
			write_text_to_log_file("For asc of bathy file please specify grdalpha in the BG_param.txt (default 0)");
		}

		if (XParam.spherical < 1)
		{
			XParam.delta = XParam.dx;
			XParam.grdalpha = XParam.grdalpha*pi / 180.0; // grid rotation
			
		}
		else
		{
			XParam.delta = XParam.dx * XParam.Radius*pi / 180.0;
			printf("Using spherical coordinate; delta=%f rad\n", XParam.delta);
			write_text_to_log_file("Using spherical coordinate; delta=" + std::to_string(XParam.delta));
			if (XParam.grdalpha != 0.0)
			{
				printf("grid rotation in spherical coordinate is not supported yet. grdalpha=%f rad\n", XParam.grdalpha);
				write_text_to_log_file("grid rotation in spherical coordinate is not supported yet. grdalpha=" + std::to_string(XParam.grdalpha*180.0 / pi));
			}
		}
		



													//fid = fopen(XParam.Bathymetryfile.c_str(), "r");
													//fscanf(fid, "%u\t%u\t%lf\t%*f\t%lf", &XParam.nx, &XParam.ny, &XParam.dx, &XParam.grdalpha);
		printf("nx=%d\tny=%d\tdx=%f\talpha=%f\txo=%f\tyo=%f\n", XParam.nx, XParam.ny, XParam.dx, XParam.grdalpha * 180.0 / pi,XParam.xo, XParam.yo);
		write_text_to_log_file("nx=" + std::to_string(XParam.nx) + " ny=" + std::to_string(XParam.ny) + " dx=" + std::to_string(XParam.dx) + " grdalpha=" + std::to_string(XParam.grdalpha*180.0 / pi) + " xo=" + std::to_string(XParam.xo) + " yo=" + std::to_string(XParam.yo));


		/////////////////////////////////////////////////////
		////// CHECK PARAMETER SANITY
		/////////////////////////////////////////////////////
		XParam = checkparamsanity(XParam);





	}
	else
	{
		std::cerr << "Fatal error: No bathymetry file specified. Please specify using 'bathy = Filename.bot'" << std::endl;
		write_text_to_log_file("Fatal error : No bathymetry file specified. Please specify using 'bathy = Filename.md'");
		exit(1);
	}

	//////////////////////////////////////////////////
	////// Preprare Bnd
	//////////////////////////////////////////////////

	// So far bnd are limited to be cst along an edge
	// Read Bnd file if/where needed
	printf("Reading and preparing Boundaries...");
	write_text_to_log_file("Reading and preparing Boundaries");

	std::vector<SLTS> leftWLbnd;
	std::vector<SLTS> rightWLbnd;
	std::vector<SLTS> topWLbnd;
	std::vector<SLTS> botWLbnd;

	if (!XParam.leftbndfile.empty())
	{
		leftWLbnd = readWLfile(XParam.leftbndfile);
		
	}
	if (!XParam.rightbndfile.empty())
	{
		rightWLbnd = readWLfile(XParam.rightbndfile);
	}
	if (!XParam.topbndfile.empty())
	{
		topWLbnd = readWLfile(XParam.topbndfile);
	}
	if (!XParam.botbndfile.empty())
	{
		botWLbnd = readWLfile(XParam.botbndfile);
	}

	XParam.endtime = setendtime(XParam, leftWLbnd, rightWLbnd, topWLbnd, botWLbnd);


	printf("...done!\n");
	write_text_to_log_file("Done Reading and preparing Boundaries");

	XParam.dt = 0.0;// Will be resolved in update

	////////////////////////////////////////////////
	///// Allocate memory on CPU
	////////////////////////////////////////////////

	printf("Allocate CPU memory...");
	write_text_to_log_file("Allocate CPU memory...");

	int nx = XParam.nx;
	int ny = XParam.ny;


	if (XParam.doubleprecision == 1 || XParam.spherical == 1)
	{
		//allocate double *arrays
		Allocate1CPU(nx, ny, zb_d);
		Allocate4CPU(nx, ny, zs_d, hh_d, uu_d, vv_d);
		Allocate4CPU(nx, ny, zso_d, hho_d, uuo_d, vvo_d);
		Allocate4CPU(nx, ny, dzsdx_d, dhdx_d, dudx_d, dvdx_d);
		Allocate4CPU(nx, ny, dzsdy_d, dhdy_d, dudy_d, dvdy_d);

		Allocate4CPU(nx, ny, Su_d, Sv_d, Fhu_d, Fhv_d);
		Allocate4CPU(nx, ny, Fqux_d, Fquy_d, Fqvx_d, Fqvy_d);

		Allocate4CPU(nx, ny, dh_d, dhu_d, dhv_d, dummy_d);


		//also allocate dummy as a float * to ease some data reading
		Allocate1CPU(nx, ny, dummy);

		//not allocating below may be usefull

		if (XParam.outhhmax == 1)
		{
			Allocate1CPU(nx, ny, hhmax_d);
		}
		if (XParam.outuumax == 1)
		{
			Allocate1CPU(nx, ny, uumax_d);
		}
		if (XParam.outvvmax == 1)
		{
			Allocate1CPU(nx, ny, vvmax_d);
		}
		if (XParam.outzsmax == 1)
		{
			Allocate1CPU(nx, ny, zsmax_d);
		}

		if (XParam.outhhmean == 1)
		{
			Allocate1CPU(nx, ny, hhmean_d);
		}
		if (XParam.outzsmean == 1)
		{
			Allocate1CPU(nx, ny, zsmean_d);
		}
		if (XParam.outuumean == 1)
		{
			Allocate1CPU(nx, ny, uumean_d);
		}
		if (XParam.outvvmean == 1)
		{
			Allocate1CPU(nx, ny, vvmean_d);
		}

		if (XParam.outvort == 1)
		{
			Allocate1CPU(nx, ny, vort);
		}

	}
	else
	{
		// allocate float *arrays (same template functions but different pointers)
		Allocate1CPU(nx, ny, zb);
		Allocate4CPU(nx, ny, zs, hh, uu, vv);
		Allocate4CPU(nx, ny, zso, hho, uuo, vvo);
		Allocate4CPU(nx, ny, dzsdx, dhdx, dudx, dvdx);
		Allocate4CPU(nx, ny, dzsdy, dhdy, dudy, dvdy);

		Allocate4CPU(nx, ny, Su, Sv, Fhu, Fhv);
		Allocate4CPU(nx, ny, Fqux, Fquy, Fqvx, Fqvy);

		Allocate4CPU(nx, ny, dh, dhu, dhv, dummy);

		//not allocating below may be usefull

		if (XParam.outhhmax == 1)
		{
			Allocate1CPU(nx, ny, hhmax);
		}
		if (XParam.outuumax == 1)
		{
			Allocate1CPU(nx, ny, uumax);
		}
		if (XParam.outvvmax == 1)
		{
			Allocate1CPU(nx, ny, vvmax);
		}
		if (XParam.outzsmax == 1)
		{
			Allocate1CPU(nx, ny, zsmax);
		}

		if (XParam.outhhmean == 1)
		{
			Allocate1CPU(nx, ny, hhmean);
		}
		if (XParam.outzsmean == 1)
		{
			Allocate1CPU(nx, ny, zsmean);
		}
		if (XParam.outuumean == 1)
		{
			Allocate1CPU(nx, ny, uumean);
		}
		if (XParam.outvvmean == 1)
		{
			Allocate1CPU(nx, ny, vvmean);
		}

		if (XParam.outvort == 1)
		{
			Allocate1CPU(nx, ny, vort);
		}

	}
	




	printf("...done!\n");
	write_text_to_log_file("Done");


	if (XParam.GPUDEVICE >= 0)
	{
		// Init GPU
		// This should be in the sanity check
		int nDevices;
		hipGetDeviceCount(&nDevices);
		hipDeviceProp_t prop;

		if (XParam.GPUDEVICE > (nDevices - 1))
		{
			// 
			XParam.GPUDEVICE = (nDevices - 1);
		}
		hipGetDeviceProperties(&prop, XParam.GPUDEVICE);
		printf("There are %d GPU devices on this machine\n", nDevices);
		printf("Using Device : %s\n", prop.name);


		write_text_to_log_file("There are " + std::to_string(nDevices) + "GPU devices on this machine");
		write_text_to_log_file("There are " + std::string(prop.name) + "GPU devices on this machine");

	}

	// Now that we checked that there was indeed a GPU available
	////////////////////////////////////////
	//////// ALLLOCATE GPU memory
	////////////////////////////////////////
	if (XParam.GPUDEVICE >= 0)
	{
		printf("Allocating GPU memory...");
		write_text_to_log_file("Allocating GPU memory");
		if (XParam.doubleprecision == 1 || XParam.spherical == 1)
		{
			Allocate1GPU(nx, ny, zb_gd);
			Allocate4GPU(nx, ny, zs_gd, hh_gd, uu_gd, vv_gd);
			Allocate4GPU(nx, ny, zso_gd, hho_gd, uuo_gd, vvo_gd);
			Allocate4GPU(nx, ny, dzsdx_gd, dhdx_gd, dudx_gd, dvdx_gd);
			Allocate4GPU(nx, ny, dzsdy_gd, dhdy_gd, dudy_gd, dvdy_gd);

			Allocate4GPU(nx, ny, Su_gd, Sv_gd, Fhu_gd, Fhv_gd);
			Allocate4GPU(nx, ny, Fqux_gd, Fquy_gd, Fqvx_gd, Fqvy_gd);

			Allocate4GPU(nx, ny, dh_gd, dhu_gd, dhv_gd, dtmax_gd);

			arrmin_d = (double *)malloc(nx*ny * sizeof(double));
			CUDA_CHECK(hipMalloc((void **)&arrmin_gd, nx*ny * sizeof(double)));
			CUDA_CHECK(hipMalloc((void **)&arrmax_gd, nx*ny * sizeof(double)));

			if (XParam.outhhmax == 1)
			{
				Allocate1GPU(nx, ny, hhmax_gd);
			}
			if (XParam.outzsmax == 1)
			{
				Allocate1GPU(nx, ny, zsmax_gd);
			}
			if (XParam.outuumax == 1)
			{
				Allocate1GPU(nx, ny, uumax_gd);
			}
			if (XParam.outvvmax == 1)
			{
				Allocate1GPU(nx, ny, vvmax_gd);
			}
			if (XParam.outhhmean == 1)
			{
				Allocate1GPU(nx, ny, hhmean_gd);
			}
			if (XParam.outzsmean == 1)
			{
				Allocate1GPU(nx, ny, zsmean_gd);
			}
			if (XParam.outuumean == 1)
			{
				Allocate1GPU(nx, ny, uumean_gd);
			}
			if (XParam.outvvmean == 1)
			{
				Allocate1GPU(nx, ny, vvmean_gd);
			}

			if (XParam.outvort == 1)
			{
				Allocate1GPU(nx, ny, vort_gd);
			}

			if (XParam.TSnodesout.size() > 0)
			{
				// Allocate mmemory to store TSoutput in between writing to disk
				int nTS = 1; // Nb of points
				int nvts = 1; // NB of variables hh, zs, uu, vv
				int nstore = 2048; //store up to 2048 pts
				TSstore_d = (double *)malloc(nTS*nvts*nstore * sizeof(double));
				CUDA_CHECK(hipMalloc((void **)&TSstore_gd, nTS*nvts*nstore * sizeof(double)));
				//Cpu part done differently because there are no latency issue (i.e. none that I care about) 

			}
		}
		else
		{
			Allocate1GPU(nx, ny, zb_g);
			Allocate4GPU(nx, ny, zs_g, hh_g, uu_g, vv_g);
			Allocate4GPU(nx, ny, zso_g, hho_g, uuo_g, vvo_g);
			Allocate4GPU(nx, ny, dzsdx_g, dhdx_g, dudx_g, dvdx_g);
			Allocate4GPU(nx, ny, dzsdy_g, dhdy_g, dudy_g, dvdy_g);

			Allocate4GPU(nx, ny, Su_g, Sv_g, Fhu_g, Fhv_g);
			Allocate4GPU(nx, ny, Fqux_g, Fquy_g, Fqvx_g, Fqvy_g);

			Allocate4GPU(nx, ny, dh_g, dhu_g, dhv_g, dtmax_g);

			arrmin = (float *)malloc(nx*ny * sizeof(float));
			CUDA_CHECK(hipMalloc((void **)&arrmin_g, nx*ny * sizeof(float)));
			CUDA_CHECK(hipMalloc((void **)&arrmax_g, nx*ny * sizeof(float)));

			if (XParam.outhhmax == 1)
			{
				CUDA_CHECK(hipMalloc((void **)&hhmax_g, nx*ny * sizeof(float)));
			}
			if (XParam.outzsmax == 1)
			{
				CUDA_CHECK(hipMalloc((void **)&zsmax_g, nx*ny * sizeof(float)));
			}
			if (XParam.outuumax == 1)
			{
				CUDA_CHECK(hipMalloc((void **)&uumax_g, nx*ny * sizeof(float)));
			}
			if (XParam.outvvmax == 1)
			{
				CUDA_CHECK(hipMalloc((void **)&vvmax_g, nx*ny * sizeof(float)));
			}
			if (XParam.outhhmean == 1)
			{
				CUDA_CHECK(hipMalloc((void **)&hhmean_g, nx*ny * sizeof(float)));
			}
			if (XParam.outzsmean == 1)
			{
				CUDA_CHECK(hipMalloc((void **)&zsmean_g, nx*ny * sizeof(float)));
			}
			if (XParam.outuumean == 1)
			{
				CUDA_CHECK(hipMalloc((void **)&uumean_g, nx*ny * sizeof(float)));
			}
			if (XParam.outvvmean == 1)
			{
				CUDA_CHECK(hipMalloc((void **)&vvmean_g, nx*ny * sizeof(float)));
			}

			if (XParam.outvort == 1)
			{
				CUDA_CHECK(hipMalloc((void **)&vort_g, nx*ny * sizeof(float)));
			}


			if (XParam.TSnodesout.size() > 0)
			{
				// Allocate mmemory to store TSoutput in between writing to disk
				int nTS = 1; // Nb of points
				int nvts = 1; // NB of variables hh, zs, uu, vv
				int nstore = 2048; //store up to 2048 pts
				TSstore = (float *)malloc(nTS*nvts*nstore * sizeof(float));
				CUDA_CHECK(hipMalloc((void **)&TSstore_g, nTS*nvts*nstore * sizeof(float)));
				//Cpu part done differently because there are no latency issue (i.e. none that I care about) 

			}
		}

		// This below was float by default and probably should remain float as long as fetched floats are readily converted to double as needed
		

		if (!XParam.leftbndfile.empty())
		{
			//leftWLbnd = readWLfile(XParam.leftbndfile);
			//Flatten bnd to copy to cuda array
			int nbndtimes = (int) leftWLbnd.size();
			int nbndvec = (int) leftWLbnd[0].wlevs.size();
			CUDA_CHECK(hipMallocArray(&leftWLS_gp, &channelDescleftbnd, nbndtimes, nbndvec));

			float * leftWLS;
			leftWLS=(float *)malloc(nbndtimes * nbndvec * sizeof(float));

			for (int ibndv = 0; ibndv < nbndvec; ibndv++)
			{
				for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
				{
					//
					leftWLS[ibndt + ibndv*nbndtimes] = leftWLbnd[ibndt].wlevs[ibndv];
				}
			}
			CUDA_CHECK(hipMemcpyToArray(leftWLS_gp, 0, 0, leftWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

			texLBND.addressMode[0] = hipAddressModeClamp;
			texLBND.addressMode[1] = hipAddressModeClamp;
			texLBND.filterMode = hipFilterModeLinear;
			texLBND.normalized = false;


			CUDA_CHECK(hipBindTextureToArray(texLBND, leftWLS_gp, channelDescleftbnd));
			free(leftWLS);

		}
		if (!XParam.rightbndfile.empty())
		{
			//leftWLbnd = readWLfile(XParam.leftbndfile);
			//Flatten bnd to copy to cuda array
			int nbndtimes = (int) rightWLbnd.size();
			int nbndvec = (int) rightWLbnd[0].wlevs.size();
			CUDA_CHECK(hipMallocArray(&rightWLS_gp, &channelDescrightbnd, nbndtimes, nbndvec));

			float * rightWLS;
			rightWLS = (float *)malloc(nbndtimes * nbndvec * sizeof(float));

			for (int ibndv = 0; ibndv < nbndvec; ibndv++)
			{
				for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
				{
					//
					rightWLS[ibndt + ibndv*nbndtimes] = rightWLbnd[ibndt].wlevs[ibndv];
				}
			}
			CUDA_CHECK(hipMemcpyToArray(rightWLS_gp, 0, 0, rightWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

			texRBND.addressMode[0] = hipAddressModeClamp;
			texRBND.addressMode[1] = hipAddressModeClamp;
			texRBND.filterMode = hipFilterModeLinear;
			texRBND.normalized = false;


			CUDA_CHECK(hipBindTextureToArray(texRBND, rightWLS_gp, channelDescrightbnd));
			free(rightWLS);

		}
		if (!XParam.topbndfile.empty())
		{
			//leftWLbnd = readWLfile(XParam.leftbndfile);
			//Flatten bnd to copy to cuda array
			int nbndtimes = (int) topWLbnd.size();
			int nbndvec = (int) topWLbnd[0].wlevs.size();
			CUDA_CHECK(hipMallocArray(&topWLS_gp, &channelDesctopbnd, nbndtimes, nbndvec));

			float * topWLS;
			topWLS = (float *)malloc(nbndtimes * nbndvec * sizeof(float));

			for (int ibndv = 0; ibndv < nbndvec; ibndv++)
			{
				for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
				{
					//
					topWLS[ibndt + ibndv*nbndtimes] = topWLbnd[ibndt].wlevs[ibndv];
				}
			}
			CUDA_CHECK(hipMemcpyToArray(topWLS_gp, 0, 0, topWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

			texTBND.addressMode[0] = hipAddressModeClamp;
			texTBND.addressMode[1] = hipAddressModeClamp;
			texTBND.filterMode = hipFilterModeLinear;
			texTBND.normalized = false;


			CUDA_CHECK(hipBindTextureToArray(texTBND, topWLS_gp, channelDesctopbnd));
			free(topWLS);

		}
		if (!XParam.botbndfile.empty())
		{
			//leftWLbnd = readWLfile(XParam.leftbndfile);
			//Flatten bnd to copy to cuda array
			int nbndtimes = (int) botWLbnd.size();
			int nbndvec = (int) botWLbnd[0].wlevs.size();
			CUDA_CHECK(hipMallocArray(&botWLS_gp, &channelDescbotbnd, nbndtimes, nbndvec));

			float * botWLS;
			botWLS = (float *)malloc(nbndtimes * nbndvec * sizeof(float));

			for (int ibndv = 0; ibndv < nbndvec; ibndv++)
			{
				for (int ibndt = 0; ibndt < nbndtimes; ibndt++)
				{
					//
					botWLS[ibndt + ibndv*nbndtimes] = botWLbnd[ibndt].wlevs[ibndv];
				}
			}
			CUDA_CHECK(hipMemcpyToArray(botWLS_gp, 0, 0, botWLS, nbndtimes * nbndvec * sizeof(float), hipMemcpyHostToDevice));

			texBBND.addressMode[0] = hipAddressModeClamp;
			texBBND.addressMode[1] = hipAddressModeClamp;
			texBBND.filterMode = hipFilterModeLinear;
			texBBND.normalized = false;


			CUDA_CHECK(hipBindTextureToArray(texBBND, botWLS_gp, channelDescbotbnd));
			free(botWLS);

		}
		printf("Done\n");
		write_text_to_log_file("Done");

	}

	printf("Read Bathy data...");
	write_text_to_log_file("Read Bathy data");

	if (bathyext.compare("md") == 0)
	{
		readbathy(XParam.Bathymetryfile, dummy);
	}
	if (bathyext.compare("nc") == 0)
	{
		readnczb(XParam.nx, XParam.ny, XParam.Bathymetryfile, dummy);
	}
	if (bathyext.compare("bot") == 0 || bathyext.compare("dep") == 0)
	{
		readXBbathy(XParam.Bathymetryfile, XParam.nx, XParam.ny, dummy);
	}
	if (bathyext.compare("asc") == 0)
	{
		//
		readbathyASCzb(XParam.Bathymetryfile, XParam.nx, XParam.ny, dummy);
	}


	
	//printf("%f\n", zb[0]);
	//printf("%f\n", zb[(nx - 1) + (0)*nx]);
	//printf("%f\n", zb[(0) + (ny-1)*nx]);
	//printf("%f\n", zb[(nx - 1) + (ny - 1)*nx]);
	

	//init variables
	if (XParam.posdown == 1)
	{
		printf("Bathy data is positive down...correcting ...");
		write_text_to_log_file("Bathy data is positive down...correcting");
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				dummy[i + j*nx] = dummy[i + j*nx] * -1.0f;
				//printf("%f\n", zb[i + (j)*nx]);
				
			}
		}
	}
	// Copy dummy to zb
	if (XParam.doubleprecision == 1 || XParam.spherical == 1)
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				zb_d[i + j*nx] = dummy[i + j*nx] * 1.0;
			}
		}
	}
	else
	{
		for (int j = 0; j < ny; j++)
		{
			for (int i = 0; i < nx; i++)
			{
				zb[i + j*nx] = dummy[i + j*nx];
			}
		}
	}


	printf("Done\n");
	write_text_to_log_file("Done");

	// set grid edges. this is necessary for boundary conditions to work
	//could be more efficient
	if (XParam.doubleprecision == 1 || XParam.spherical == 1)
	{
		setedges(nx, ny, zb_d);
	}
	else 
	{
		setedges(nx, ny, zb);
	}
	

	/////////////////////////////////////////////////////
	// Initial Condition
	/////////////////////////////////////////////////////
	printf("Initial condition: ");
	write_text_to_log_file("Initial condition:");

	int hotstartsucess = 0;
	if (!XParam.hotstartfile.empty())
	{
		// hotstart
		printf("Hotstart "); 
		write_text_to_log_file("Hotstart");
		if (XParam.doubleprecision == 1 || XParam.spherical == 1)
		{
			hotstartsucess = readhotstartfileD(XParam, zs_d, zb_d, hh_d, uu_d, vv_d);
		}
		else
		{
			hotstartsucess = readhotstartfile(XParam, zs, zb, hh, uu, vv);
		}
		
		if (hotstartsucess == 0)
		{
			printf("Failed...  ");
			write_text_to_log_file("Hotstart failed switching to cold start");
		}
	}
	if (XParam.hotstartfile.empty() || hotstartsucess == 0)
	{
		printf("Cold start  ");
		write_text_to_log_file("Cold start");
		//Cold start
		// 2 options: 
		//		(1) if zsinit is set, then apply zsinit everywhere
		//		(2) zsinit is not set so interpolate from boundaries. (if no boundaries were specified set zsinit to zeros and apply case (1))

		Param defaultParam;
		//!leftWLbnd.empty()
		
		//case 2b (i.e. zsinint and no boundaries were specified)
		if ((abs(XParam.zsinit - defaultParam.zsinit) <= epsilon) && (leftWLbnd.empty() && rightWLbnd.empty() && topWLbnd.empty() && botWLbnd.empty()) ) //zsinit is default
		{
			XParam.zsinit = 0.0; // better default value
		}

		//case(1)
		if (abs(XParam.zsinit - defaultParam.zsinit) > epsilon) // apply specified zsinit
		{
			if (XParam.doubleprecision == 1 || XParam.spherical == 1)
			{
				for (int j = 0; j < ny; j++)
				{
					for (int i = 0; i < nx; i++)
					{

						uu_d[i + j*nx] = 0.0;
						vv_d[i + j*nx] = 0.0;
						//zb[i + j*nx] = 0.0f;
						zs_d[i + j*nx] = max(XParam.zsinit, zb_d[i + j*nx]);
						//if (i >= 64 && i < 82)
						//{
						//	zs[i + j*nx] = max(zsbnd+0.2f, zb[i + j*nx]);
						//}
						hh_d[i + j*nx] = max(zs_d[i + j*nx] - zb_d[i + j*nx], XParam.eps);


					}
				}
			}
			else
			{
				for (int j = 0; j < ny; j++)
				{
					for (int i = 0; i < nx; i++)
					{

						uu[i + j*nx] = 0.0f;
						vv[i + j*nx] = 0.0f;
						//zb[i + j*nx] = 0.0f;
						zs[i + j*nx] = max((float)XParam.zsinit, zb[i + j*nx]);
						//if (i >= 64 && i < 82)
						//{
						//	zs[i + j*nx] = max(zsbnd+0.2f, zb[i + j*nx]);
						//}
						hh[i + j*nx] = max(zs[i + j*nx] - zb[i + j*nx], (float)XParam.eps);


					}
				}
			}

		}
		else // lukewarm start i.e. bilinear interpolation of zs
		{
			double zsleft = 0.0;
			double zsright = 0.0;
			double zstop = 0.0;
			double zsbot = 0.0;
			double zsbnd = 0.0;

			double distleft, distright, disttop, distbot;

			double lefthere = 0.0;
			double righthere = 0.0;
			double tophere = 0.0;
			double bothere = 0.0;


			for (int j = 0; j < ny; j++)
			{
				disttop = max((double)(ny - 1) - j, 0.1);
				
				distbot = max((double) j, 0.1);

				if (XParam.left == 1 && !leftWLbnd.empty())
				{
					lefthere = 1.0;
					int SLstepinbnd = 1;



					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = leftWLbnd[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = leftWLbnd[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<double> zsbndvec;
					for (int n = 0; n < leftWLbnd[SLstepinbnd].wlevs.size(); n++)
					{
						zsbndvec.push_back( interptime(leftWLbnd[SLstepinbnd].wlevs[n], leftWLbnd[SLstepinbnd - 1].wlevs[n], leftWLbnd[SLstepinbnd].time - leftWLbnd[SLstepinbnd - 1].time, XParam.totaltime - leftWLbnd[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zsleft = zsbndvec[0];
					}
					else
					{
						int iprev = min(max((int)ceil(j / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zsleft =  interptime(zsbndvec[inext], zsbndvec[iprev], (double)(inext - iprev), (double)(j - iprev));
					}

				}
				
				if (XParam.right == 1 && !rightWLbnd.empty())
				{
					int SLstepinbnd = 1;
					righthere = 1.0;


					// Do this for all the corners
					//Needs limiter in case WLbnd is empty
					double difft = rightWLbnd[SLstepinbnd].time - XParam.totaltime;

					while (difft < 0.0)
					{
						SLstepinbnd++;
						difft = rightWLbnd[SLstepinbnd].time - XParam.totaltime;
					}
					std::vector<double> zsbndvec;
					for (int n = 0; n < rightWLbnd[SLstepinbnd].wlevs.size(); n++)
					{
						zsbndvec.push_back( interptime(rightWLbnd[SLstepinbnd].wlevs[n], rightWLbnd[SLstepinbnd - 1].wlevs[n], rightWLbnd[SLstepinbnd].time - rightWLbnd[SLstepinbnd - 1].time, XParam.totaltime - rightWLbnd[SLstepinbnd - 1].time));

					}
					if (zsbndvec.size() == 1)
					{
						zsright = zsbndvec[0];
					}
					else
					{
						int iprev = min(max((int)ceil(j / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
						int inext = iprev + 1;
						// here interp time is used to interpolate to the right node rather than in time...
						zsright = interptime(zsbndvec[inext], zsbndvec[iprev], (double)(inext - iprev), (double)(j - iprev));
					}


				}
				
				
				
				

				for (int i = 0; i < nx; i++)
				{
					distleft = max((double)i,0.1);
					distright = max((double)(nx - 1) - i, 0.1);

					if (XParam.bot == 1 && !botWLbnd.empty())
					{
						int SLstepinbnd = 1;
						bothere = 1.0;




						// Do this for all the corners
						//Needs limiter in case WLbnd is empty
						double difft = botWLbnd[SLstepinbnd].time - XParam.totaltime;

						while (difft < 0.0)
						{
							SLstepinbnd++;
							difft = botWLbnd[SLstepinbnd].time - XParam.totaltime;
						}
						std::vector<double> zsbndvec;
						for (int n = 0; n < botWLbnd[SLstepinbnd].wlevs.size(); n++)
						{
							zsbndvec.push_back(interptime(botWLbnd[SLstepinbnd].wlevs[n], botWLbnd[SLstepinbnd - 1].wlevs[n], botWLbnd[SLstepinbnd].time - botWLbnd[SLstepinbnd - 1].time, XParam.totaltime - botWLbnd[SLstepinbnd - 1].time));

						}
						if (zsbndvec.size() == 1)
						{
							zsbot = zsbndvec[0];
						}
						else
						{
							int iprev = min(max((int)ceil(i / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
							int inext = iprev + 1;
							// here interp time is used to interpolate to the right node rather than in time...
							zsbot =  interptime(zsbndvec[inext], zsbndvec[iprev], (double)(inext - iprev), (double)(i - iprev));
						}

					}
					if (XParam.top == 1 && !topWLbnd.empty())
					{
						int SLstepinbnd = 1;
						tophere = 1.0;




						// Do this for all the corners
						//Needs limiter in case WLbnd is empty
						double difft = topWLbnd[SLstepinbnd].time - XParam.totaltime;

						while (difft < 0.0)
						{
							SLstepinbnd++;
							difft = topWLbnd[SLstepinbnd].time - XParam.totaltime;
						}
						std::vector<double> zsbndvec;
						for (int n = 0; n < topWLbnd[SLstepinbnd].wlevs.size(); n++)
						{
							zsbndvec.push_back( interptime(topWLbnd[SLstepinbnd].wlevs[n], topWLbnd[SLstepinbnd - 1].wlevs[n], topWLbnd[SLstepinbnd].time - topWLbnd[SLstepinbnd - 1].time, XParam.totaltime - topWLbnd[SLstepinbnd - 1].time));

						}
						if (zsbndvec.size() == 1)
						{
							zstop = zsbndvec[0];
						}
						else
						{
							int iprev = min(max((int)ceil(i / (1 / (zsbndvec.size() - 1))), 0), (int)zsbndvec.size() - 2);
							int inext = iprev + 1;
							// here interp time is used to interpolate to the right node rather than in time...
							zstop =  interptime(zsbndvec[inext], zsbndvec[iprev], (double)(inext - iprev), (double)(i - iprev));
						}

					}
				
										

					//if (XParam.top == 1 && !topWLbnd.empty() && XParam.bot == 1 && !botWLbnd.empty() && XParam.left == 1 && !leftWLbnd.empty() && XParam.right == 1 && !rightWLbnd.empty())
					//{
					//	zsbnd = (zsleft*(1 / i) + zsright * 1 / (nx - i) + zsbot * 1 / j + zstop * 1 / (ny - j)) / ((1 / i) + 1 / (nx - i) + 1 / j + 1 / (ny - j));
					//}
					
					zsbnd = ((zsleft * 1 / distleft)*lefthere + (zsright * 1 / distright)*righthere + (zstop * 1 / disttop)*tophere + (zsbot * 1 / distbot)*bothere) / ((1 / distleft)*lefthere + (1 / distright)*righthere + (1 / disttop)*tophere + (1 / distbot)*bothere);
					
					if (XParam.doubleprecision == 1 || XParam.spherical == 1)
					{
						zs_d[i + j*nx] = max(zsbnd, zb_d[i + j*nx]);
						hh_d[i + j*nx] = max(zs_d[i + j*nx] - zb_d[i + j*nx], XParam.eps);
						uu_d[i + j*nx] = 0.0;
						vv_d[i + j*nx] = 0.0;
					}
					else
					{
						zs[i + j*nx] = max((float)zsbnd, zb[i + j*nx]);
						hh[i + j*nx] = max(zs[i + j*nx] - zb[i + j*nx], (float)XParam.eps);
						uu[i + j*nx] = 0.0f;
						vv[i + j*nx] = 0.0f;
					}

				}
			}


		}

		



		
	}
	printf("done \n  ");
	write_text_to_log_file("Done");
	// Below is not succint but way faster than one loop that checks teh if statemenst each time
	if (XParam.doubleprecision == 1 || XParam.spherical == 1)
	{
		if (XParam.outhhmax == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					hhmax_d[i + j*nx] = hh_d[i + j*nx];
				}
			}
		}

		if (XParam.outhhmean == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					hhmean_d[i + j*nx] = 0.0;
				}
			}
		}
		if (XParam.outzsmax == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					zsmax_d[i + j*nx] = zs_d[i + j*nx];
				}
			}
		}

		if (XParam.outzsmean == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					zsmean_d[i + j*nx] = 0.0;
				}
			}
		}

		if (XParam.outuumax == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					uumax_d[i + j*nx] = uu_d[i + j*nx];
				}
			}
		}

		if (XParam.outuumean == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					uumean_d[i + j*nx] = 0.0;
				}
			}
		}
		if (XParam.outvvmax == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					vvmax_d[i + j*nx] = vv_d[i + j*nx];
				}
			}
		}

		if (XParam.outvvmean == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					vvmean_d[i + j*nx] = 0.0;
				}
			}
		}
		if (XParam.outvort == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					vort_d[i + j*nx] = 0.0;
				}
			}
		}
	}
	else //Using Float *
	{
		
		if (XParam.outhhmax == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					hhmax[i + j*nx] = hh[i + j*nx];
				}
			}
		}

		if (XParam.outhhmean == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					hhmean[i + j*nx] = 0.0;
				}
			}
		}
		if (XParam.outzsmax == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					zsmax[i + j*nx] = zs[i + j*nx];
				}
			}
		}

		if (XParam.outzsmean == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					zsmean[i + j*nx] = 0.0;
				}
			}
		}

		if (XParam.outuumax == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					uumax[i + j*nx] = uu[i + j*nx];
				}
			}
		}

		if (XParam.outuumean == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					uumean[i + j*nx] = 0.0;
				}
			}
		}
		if (XParam.outvvmax == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					vvmax[i + j*nx] = vv[i + j*nx];
				}
			}
		}

		if (XParam.outvvmean == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					vvmean[i + j*nx] = 0.0;
				}
			}
		}
		if (XParam.outvort == 1)
		{
			for (int j = 0; j < ny; j++)
			{
				for (int i = 0; i < nx; i++)
				{
					vort[i + j*nx] = 0.0;
				}
			}
		}
	}
	if (XParam.GPUDEVICE >= 0)
	{
		printf("Init data on GPU ");
		write_text_to_log_file("Init data on GPU ");

		dim3 blockDim(16, 16, 1);// The grid has a better ocupancy when the size is a factor of 16 on both x and y
		dim3 gridDim(ceil((nx*1.0f) / blockDim.x), ceil((ny*1.0f) / blockDim.y), 1);

		if (XParam.doubleprecision == 1 || XParam.spherical == 1)
		{
			CUDA_CHECK(hipMemcpy(zb_gd, zb_d, nx*ny * sizeof(double), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(hh_gd, hh_d, nx*ny * sizeof(double), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(uu_gd, uu_d, nx*ny * sizeof(double), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(vv_gd, vv_d, nx*ny * sizeof(double), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(zs_gd, zs_d, nx*ny * sizeof(double), hipMemcpyHostToDevice));
			initdtmax << <gridDim, blockDim, 0 >> >(nx, ny, epsilon, dtmax_gd);
		}
		else
		{
			CUDA_CHECK(hipMemcpy(zb_g, zb, nx*ny * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(hh_g, hh, nx*ny * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(uu_g, uu, nx*ny * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(vv_g, vv, nx*ny * sizeof(float), hipMemcpyHostToDevice));
			CUDA_CHECK(hipMemcpy(zs_g, zs, nx*ny * sizeof(float), hipMemcpyHostToDevice));
			initdtmax << <gridDim, blockDim, 0 >> >(nx, ny, (float)epsilon, dtmax_g);
		}
		




		

		
		CUDA_CHECK(hipDeviceSynchronize());
		printf("...Done\n ");
		write_text_to_log_file("Done ");

	}

	// Here map array to their name as a string. it makes it super easy to convert user define variables to the array it represents.
	// COul add more to output gradients etc...
	OutputVarMapCPU["zb"] = zb;
	OutputVarMapCPUD["zb"] = zb_d;
	OutputVarMapGPU["zb"] = zb_g;
	OutputVarMapGPUD["zb"] = zb_gd;
	OutputVarMaplen["zb"] = nx*ny;

	OutputVarMapCPU["uu"] = uu;
	OutputVarMapCPUD["uu"] = uu_d;
	OutputVarMapGPU["uu"] = uu_g;
	OutputVarMapGPUD["uu"] = uu_gd;
	OutputVarMaplen["uu"] = nx*ny;

	OutputVarMapCPU["vv"] = vv;
	OutputVarMapCPUD["vv"] = vv_d;
	OutputVarMapGPU["vv"] = vv_g;
	OutputVarMapGPUD["vv"] = vv_gd;
	OutputVarMaplen["vv"] = nx*ny;

	OutputVarMapCPU["zs"] = zs;
	OutputVarMapCPUD["zs"] = zs_d;
	OutputVarMapGPU["zs"] = zs_g;
	OutputVarMapGPUD["zs"] = zs_gd;
	OutputVarMaplen["zs"] = nx*ny;

	OutputVarMapCPU["hh"] = hh;
	OutputVarMapCPUD["hh"] = hh_d;
	OutputVarMapGPU["hh"] = hh_g;
	OutputVarMapGPUD["hh"] = hh_gd;
	OutputVarMaplen["hh"] = nx*ny;

	OutputVarMapCPU["hhmean"] = hhmean;
	OutputVarMapCPUD["hhmean"] = hhmean_d;
	OutputVarMapGPU["hhmean"] = hhmean_g;
	OutputVarMapGPUD["hhmean"] = hhmean_gd;
	OutputVarMaplen["hhmean"] = nx*ny;

	OutputVarMapCPU["hhmax"] = hhmax;
	OutputVarMapCPUD["hhmax"] = hhmax_d;
	OutputVarMapGPU["hhmax"] = hhmax_g;
	OutputVarMapGPUD["hhmax"] = hhmax_gd;
	OutputVarMaplen["hhmax"] = nx*ny;

	OutputVarMapCPU["zsmean"] = zsmean;
	OutputVarMapCPUD["zsmean"] = zsmean_d;
	OutputVarMapGPU["zsmean"] = zsmean_g;
	OutputVarMapGPUD["zsmean"] = zsmean_gd;
	OutputVarMaplen["zsmean"] = nx*ny;

	OutputVarMapCPU["zsmax"] = zsmax;
	OutputVarMapCPUD["zsmax"] = zsmax_d;
	OutputVarMapGPU["zsmax"] = zsmax_g;
	OutputVarMapGPUD["zsmax"] = zsmax_gd;
	OutputVarMaplen["zsmax"] = nx*ny;

	OutputVarMapCPU["uumean"] = uumean;
	OutputVarMapCPUD["uumean"] = uumean_d;
	OutputVarMapGPU["uumean"] = uumean_g;
	OutputVarMapGPUD["uumean"] = uumean_gd;
	OutputVarMaplen["uumean"] = nx*ny;

	OutputVarMapCPU["uumax"] = uumax;
	OutputVarMapCPUD["uumax"] = uumax_d;
	OutputVarMapGPU["uumax"] = uumax_g;
	OutputVarMapGPUD["uumax"] = uumax_gd;
	OutputVarMaplen["uumax"] = nx*ny;

	OutputVarMapCPU["vvmean"] = vvmean;
	OutputVarMapCPUD["vvmean"] = vvmean_d;
	OutputVarMapGPU["vvmean"] = vvmean_g;
	OutputVarMapGPUD["vvmean"] = vvmean_gd;
	OutputVarMaplen["vvmean"] = nx*ny;

	OutputVarMapCPU["vvmax"] = vvmax;
	OutputVarMapCPUD["vvmax"] = vvmax_d;
	OutputVarMapGPU["vvmax"] = vvmax_g;
	OutputVarMapGPUD["vvmax"] = vvmax_gd;
	OutputVarMaplen["vvmax"] = nx*ny;

	OutputVarMapCPU["vort"] = vort;
	OutputVarMapCPUD["vort"] = vort_d;
	OutputVarMapGPU["vort"] = vort_g;
	OutputVarMapGPUD["vort"] = vort_gd;
	OutputVarMaplen["vort"] = nx*ny;


	printf("Create netCDF output file ");
	write_text_to_log_file("Create netCDF output file ");
	//create nc file with no variables
	XParam=creatncfileUD(XParam);
	for (int ivar = 0; ivar < XParam.outvars.size(); ivar++)
	{
		//Create definition for each variable and store it
		if (XParam.doubleprecision == 1 || XParam.spherical == 1)
		{
			defncvarD(XParam.outfile, XParam.smallnc, XParam.scalefactor, XParam.addoffset, nx, ny, XParam.outvars[ivar], 3, OutputVarMapCPUD[XParam.outvars[ivar]]);
		}
		else
		{
			defncvar(XParam.outfile, XParam.smallnc, XParam.scalefactor, XParam.addoffset, nx, ny, XParam.outvars[ivar], 3, OutputVarMapCPU[XParam.outvars[ivar]]);
		}
		
	}
	//create2dnc(nx, ny, dx, dx, 0.0, xx, yy, hh);

	printf("done \n ");
	write_text_to_log_file("Done ");

	
	SaveParamtolog(XParam);


	printf("Starting Model.\n ");
	write_text_to_log_file("Starting Model. ");

	if (XParam.GPUDEVICE >= 0)
	{
		mainloopGPU(XParam, leftWLbnd, rightWLbnd, topWLbnd, botWLbnd);
		//checkloopGPU(XParam);
			
	}
	else
	{
		mainloopCPU(XParam, leftWLbnd, rightWLbnd, topWLbnd, botWLbnd);
	}

	
	



	XParam.endcputime = clock();
	printf("End Computation \n");
	write_text_to_log_file("End Computation" );

	printf("Total runtime= %d  seconds\n", (XParam.endcputime - XParam.startcputime) / CLOCKS_PER_SEC);
	write_text_to_log_file("Total runtime= " + std::to_string((XParam.endcputime - XParam.startcputime) / CLOCKS_PER_SEC) + "  seconds" );

	if (XParam.doubleprecision == 1 || XParam.spherical == 1)
	{
		free(hh_d);
		free(uu_d);
		free(vv_d);
		free(zb_d);
		free(zs_d);

		free(hho_d);
		free(uuo_d);
		free(vvo_d);
		free(zso_d);

		free(dhdx_d);
		free(dhdy_d);
		free(dudx_d);
		free(dudy_d);
		free(dvdx_d);
		free(dvdy_d);

		free(dzsdx_d);
		free(dzsdy_d);

		free(Su_d);
		free(Sv_d);
		free(Fqux_d);
		free(Fquy_d);
		free(Fqvx_d);
		free(Fqvy_d);
		free(Fhu_d);
		free(Fhv_d);

		free(dh_d);
		free(dhu_d);
		free(dhv_d);

		if (XParam.outhhmax == 1)
		{
			free(hhmax_d);
		}

		if (XParam.outzsmax == 1)
		{
			free(zsmax_d);
		}
		if (XParam.outuumax == 1)
		{
			free(uumax_d);
		}
		if (XParam.outvvmax == 1)
		{
			free(vvmax_d);
		}
		if (XParam.outhhmean == 1)
		{
			free(hhmean_d);
		}
		if (XParam.outzsmean == 1)
		{
			free(zsmean_d);
		}
		if (XParam.outuumean == 1)
		{
			free(uumean_d);
		}
		if (XParam.outvvmean == 1)
		{
			free(vvmax_d);
		}

		if (XParam.outvort == 1)
		{
			free(vort_d);
		}

		if (XParam.GPUDEVICE >= 0)
		{
			hipFree(hh_gd);
			hipFree(uu_gd);
			hipFree(vv_gd);
			hipFree(zb_gd);
			hipFree(zs_gd);

			hipFree(hho_gd);
			hipFree(uuo_gd);
			hipFree(vvo_gd);
			hipFree(zso_gd);

			hipFree(dhdx_gd);
			hipFree(dhdy_gd);
			hipFree(dudx_gd);
			hipFree(dudy_gd);
			hipFree(dvdx_gd);
			hipFree(dvdy_gd);

			hipFree(dzsdx_gd);
			hipFree(dzsdy_gd);

			hipFree(Su_gd);
			hipFree(Sv_gd);
			hipFree(Fqux_gd);
			hipFree(Fquy_gd);
			hipFree(Fqvx_gd);
			hipFree(Fqvy_gd);
			hipFree(Fhu_gd);
			hipFree(Fhv_gd);

			hipFree(dh_gd);
			hipFree(dhu_gd);
			hipFree(dhv_gd);

			hipFree(dtmax_gd);


			hipFree(arrmin_gd);
			hipFree(arrmax_gd);

			if (XParam.outhhmax == 1)
			{
				hipFree(hhmax_gd);
			}

			if (XParam.outzsmax == 1)
			{
				hipFree(zsmax_gd);
			}
			if (XParam.outuumax == 1)
			{
				hipFree(uumax_gd);
			}
			if (XParam.outvvmax == 1)
			{
				hipFree(vvmax_gd);
			}
			if (XParam.outhhmean == 1)
			{
				hipFree(hhmean_gd);
			}
			if (XParam.outzsmean == 1)
			{
				hipFree(zsmean_gd);
			}
			if (XParam.outuumean == 1)
			{
				hipFree(uumean_gd);
			}
			if (XParam.outvvmean == 1)
			{
				hipFree(vvmax_gd);
			}

			if (XParam.outvort == 1)
			{
				hipFree(vort_gd);
			}

			hipDeviceReset();

		}
	}
	else
	{
		free(hh);
		free(uu);
		free(vv);
		free(zb);
		free(zs);

		free(hho);
		free(uuo);
		free(vvo);
		free(zso);

		free(dhdx);
		free(dhdy);
		free(dudx);
		free(dudy);
		free(dvdx);
		free(dvdy);

		free(dzsdx);
		free(dzsdy);

		free(Su);
		free(Sv);
		free(Fqux);
		free(Fquy);
		free(Fqvx);
		free(Fqvy);
		free(Fhu);
		free(Fhv);

		free(dh);
		free(dhu);
		free(dhv);

		if (XParam.outhhmax == 1)
		{
			free(hhmax);
		}

		if (XParam.outzsmax == 1)
		{
			free(zsmax);
		}
		if (XParam.outuumax == 1)
		{
			free(uumax);
		}
		if (XParam.outvvmax == 1)
		{
			free(vvmax);
		}
		if (XParam.outhhmean == 1)
		{
			free(hhmean);
		}
		if (XParam.outzsmean == 1)
		{
			free(zsmean);
		}
		if (XParam.outuumean == 1)
		{
			free(uumean);
		}
		if (XParam.outvvmean == 1)
		{
			free(vvmax);
		}

		if (XParam.outvort == 1)
		{
			free(vort);
		}




		if (XParam.GPUDEVICE >= 0)
		{
			hipFree(hh_g);
			hipFree(uu_g);
			hipFree(vv_g);
			hipFree(zb_g);
			hipFree(zs_g);

			hipFree(hho_g);
			hipFree(uuo_g);
			hipFree(vvo_g);
			hipFree(zso_g);

			hipFree(dhdx_g);
			hipFree(dhdy_g);
			hipFree(dudx_g);
			hipFree(dudy_g);
			hipFree(dvdx_g);
			hipFree(dvdy_g);

			hipFree(dzsdx_g);
			hipFree(dzsdy_g);

			hipFree(Su_g);
			hipFree(Sv_g);
			hipFree(Fqux_g);
			hipFree(Fquy_g);
			hipFree(Fqvx_g);
			hipFree(Fqvy_g);
			hipFree(Fhu_g);
			hipFree(Fhv_g);

			hipFree(dh_g);
			hipFree(dhu_g);
			hipFree(dhv_g);

			hipFree(dtmax_g);


			hipFree(arrmin_g);
			hipFree(arrmax_g);

			if (XParam.outhhmax == 1)
			{
				hipFree(hhmax_g);
			}

			if (XParam.outzsmax == 1)
			{
				hipFree(zsmax_g);
			}
			if (XParam.outuumax == 1)
			{
				hipFree(uumax_g);
			}
			if (XParam.outvvmax == 1)
			{
				hipFree(vvmax_g);
			}
			if (XParam.outhhmean == 1)
			{
				hipFree(hhmean_g);
			}
			if (XParam.outzsmean == 1)
			{
				hipFree(zsmean_g);
			}
			if (XParam.outuumean == 1)
			{
				hipFree(uumean_g);
			}
			if (XParam.outvvmean == 1)
			{
				hipFree(vvmax_g);
			}

			if (XParam.outvort == 1)
			{
				hipFree(vort_g);
			}

			hipDeviceReset();

		}
	}


	








	exit(0);
}

